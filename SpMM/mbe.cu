#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <bits/stdc++.h>
#include <hip/hip_cooperative_groups.h>
#include <sys/ioctl.h>
using namespace std;
using namespace std::chrono;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define INF  1073741824
#define ONE  1
#define ZERO 0

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

typedef struct
{
	unordered_set<int> L;
	unordered_set<int> R;
} Biclique;

void my_memset(int *SA, int val, int len) {
    for (int i = 0; i < len; i++)
        SA[i] = val;
}

void my_memset_order(int *SA, int val_start, int val_end) {
    for (int i = val_start; i < val_end; i++)
        SA[i - val_start] = i;
}

void maximal_bic_enum(int *NUM_NODES, int *NUM_EDGES, Node *node, int *edge,
                      int *L, int *R, int *P, int *Q, int lvl) {
    // int    v[*NUM_NODES]; my_memset(&v   ,         -1, *NUM_NODES);
    int    x[*NUM_NODES]; my_memset(x   ,         -1, *NUM_NODES);
    int  u2L[*NUM_NODES]; my_memset_order(u2L, 0, *NUM_NODES);
    int L_lp[*NUM_NODES]; my_memset(L_lp, *NUM_NODES, *NUM_NODES);
    int R_lp[*NUM_NODES]; my_memset(R_lp,          0, *NUM_NODES);
    int P_lp[*NUM_NODES]; my_memset(P_lp, *NUM_NODES, *NUM_NODES);
    int Q_lp[*NUM_NODES]; my_memset(Q_lp,          0, *NUM_NODES);
    vector<Biclique> maximal_bicliques;
    
    for (lvl = 0; lvl >= 0; ) {

        //// printf("lvl: %d\n", lvl);

        // int *v_cur    = &(   v[lvl]);
        int *x_cur    = &(   x[lvl]);
        int *L_lp_cur = &(L_lp[lvl]), *L_lp_nxt = &(L_lp[lvl+1]);
        int *R_lp_cur = &(R_lp[lvl]), *R_lp_nxt = &(R_lp[lvl+1]);
        int *P_lp_cur = &(P_lp[lvl]), *P_lp_nxt = &(P_lp[lvl+1]);
        int *Q_lp_cur = &(Q_lp[lvl]), *Q_lp_nxt = &(Q_lp[lvl+1]);
        bool is_recursive = false;

        // while P ≠ ∅ do
        while (*P_lp_cur != 0) {

            //// string tab_level(lvl << 3, ' ');
            //// printf("\n%sL:", tab_level.c_str());
            //// for (int i = 0; i < *L_lp_cur; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n%sR:", tab_level.c_str());
            //// for (int i = 0; i < *R_lp_cur; i++)
            ////     printf(" %d", R[i]);
            //// printf("\n%sP:", tab_level.c_str());
            //// for (int i = 0; i < *P_lp_cur; i++)
            ////     printf(" %d", P[i]);
            //// printf("\n%sQ:", tab_level.c_str());
            //// for (int i = 0; i < *Q_lp_cur; i++)
            ////     printf(" %d", Q[i]);

            // Select x from P;
            // P <--- P \ {x};
            *x_cur = P[--(*P_lp_cur)];
            //// printf("x: %d\n", *x_cur);
            
            // R' <--- R ∪ {x};
            *R_lp_nxt = *R_lp_cur;
            R[(*R_lp_nxt)++] = *x_cur;

            *L_lp_nxt = 0; // |L'|
            // L' <--- {u ∈ L | (u, x) ∈ E(G)};
            for (int eid = node[*x_cur].start, eid_end = eid + node[*x_cur].length; eid < eid_end; eid++) {
                int u = edge[eid];
                int l = u2L[u];
                if (l < *L_lp_cur) {
                    swap(L[(*L_lp_nxt)++], L[l]);
                    swap(u2L[L[l]], u2L[u]);
                }
            }

            //// printf("L':");
            //// for (int i = 0; i < *L_lp_nxt; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n");
            
            // P' ← ∅; Q' ← ∅;
            *P_lp_nxt = 0; *Q_lp_nxt = *Q_lp_cur;

            bool is_maximal = true;

            // foreach v ∈ Q
            for (int i = 0; i < *Q_lp_cur; i++) {

                int v = Q[i];

                int num_N_v = 0; // |N[v]|
                // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                for (int eid = node[v].start, eid_end = eid + node[v].length; eid < eid_end; eid++) {
                    int u = edge[eid];
                    int l = u2L[u];
                    if (l < *L_lp_nxt)
                        num_N_v++;
                }
                
                // if |N[v]| = |L'| then
                if (num_N_v == *L_lp_nxt) {
                    is_maximal = false;
                    break;
                }
                // // else if |N[v]| > 0 then
                // else if (num_N_v > 0)
                //     // Q' ← Q' ∪ {v};
                //     (*Q_nxt).insert(v);
                
            }

            //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

            // if is_maximal = TRUE then
            if (is_maximal == true) {

                // foreach v ∈ P do
                for (int i = 0; i < *P_lp_cur; i++) {
                    int v = P[i];

                    int num_N_v = 0; // |N[v]|
                    // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                    for (int eid = node[v].start, eid_end = eid + node[v].length; eid < eid_end; eid++) {
                        int u = edge[eid];
                        int l = u2L[u];
                        if (l < *L_lp_nxt)
                            num_N_v++;
                    }
                    
                    // if |N[v]| = |L'| then
                    if (num_N_v == *L_lp_nxt)
                        // R' ← R' ∪ {v};
                        R[(*R_lp_nxt)++] = v;
                    // else if |N[v]| > 0 then
                    else if (num_N_v > 0)
                        // P' ← P' ∪ {v};
                        swap(P[(*P_lp_nxt)++], P[i]);

                }
                
                //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

                // PRINT(L', R');
                //// printf("\n-------------- find maximal biclique --------------");
                //// printf("\nL':");
                //// for (int i = 0; i < *L_lp_nxt; i++)
                ////     printf(" %d", L[i]);
                //// printf("\nR':");
                //// for (int i = 0; i < *R_lp_nxt; i++)
                ////     printf(" %d", R[i]);
                //// printf("\n");
                //// printf("---------------------------------------------------\n");

                // save maximal bicliques
                Biclique new_maximal_bicliques;
                for (int i = 0; i < *L_lp_nxt; i++)
                    new_maximal_bicliques.L.insert(L[i]);
                for (int i = 0; i < *R_lp_nxt; i++)
                    new_maximal_bicliques.R.insert(R[i]);
                maximal_bicliques.push_back(new_maximal_bicliques);
                if (rand() % 100000 == 0) printf(".\n");

                // if P' ≠ ∅ then
                if (*P_lp_nxt != 0) {
                    // biclique_find(G, L', R', P', Q');
                    //// printf("\n往 下 安安");
                    lvl++;
                    is_recursive = true;
                    break;
                }

            }
            else {
                //// printf("\n不安安");
            }

            // Q ← Q ∪ {x};
            Q[(*Q_lp_cur)++] = *x_cur;
            //// printf("\n往 右 安安");
        }

        if (!is_recursive) {
            lvl--;
            Q[Q_lp[lvl]++] = x[lvl];
            //// printf("\n往 上 安安");
            //// printf("\n往 右 安安");
        }

    }

    printf("Find %d maximal bicliques.\n", maximal_bicliques.size());
    if (*NUM_NODES > 100) return;

    // string _ = "";
    // printf("\33[2J\33[0;0H");

    // printf("  ");
    // for (int i = 0; i < *NUM_NODES; i++)
    //     printf(" %d", i / 10);
    // printf("\n  ");
    // for (int i = 0; i < *NUM_NODES; i++)
    //     printf(" %d", i % 10);
    // printf("\n");
    // for (int i = 0; i < *NUM_NODES; i++) {
    //     bool adj_vec[*NUM_NODES] = { false };
    //     for (int j = node[i].start, j_end = j + node[i].length; j < j_end; j++)
    //         adj_vec[edge[j]] = true;
    //     printf("%d%d", i / 10, i % 10);
    //     for (int j = 0; j < *NUM_NODES; j++)
    //         printf(" %c", adj_vec[j] ? '#' : '-');
    //     printf("\n");
    // }

    // for (int i = 0, i_end = maximal_bicliques.size(); i < i_end; i++) {
    //     printf("\33[7m");
    //     for (const auto &r: maximal_bicliques[i].R)
    //         for (const auto &l: maximal_bicliques[i].L)
    //             printf("\33[%d;%dH#\n", 3 + r, 4 + l * 2);
    //     printf("\33[0m\n\33[%d;0H\n", 3 + (*NUM_NODES));
    //     if      (_ == "auto") usleep(800000);
    //     else if (_ != "exit") cin >> _;
    //     for (const auto &r: maximal_bicliques[i].R)
    //         for (const auto &l: maximal_bicliques[i].L)
    //             printf("\33[%d;%dH#\n", 3 + r, 4 + l * 2);
    // }
    // printf("\33[%d;0H\n", 3 + (*NUM_NODES));
}

int main(int argc, char* argv[])
{
    string str_dataset = argv[1];
    cout << str_dataset.substr(str_dataset.rfind('/')+1) << "\n";

    Node *node;
	int *edge;
    int *NUM_NODES, *NUM_EDGES, SOURCE, _;
    int *L, *R, *P, *Q;
    hipMallocManaged(&NUM_EDGES, sizeof(int));
    hipMallocManaged(&NUM_NODES, sizeof(int));

    ifstream fin;
    fin.open(argv[1]);
    fin >> *NUM_NODES >> *NUM_EDGES >> SOURCE;
    hipMallocManaged(&node, sizeof(Node)*(*NUM_NODES));
    hipMallocManaged(&edge, sizeof(int)*(*NUM_EDGES));
    for(int i=0;i<*NUM_NODES;i++) fin >> node[i].start >> node[i].length;
    for(int i=0;i<*NUM_EDGES;i++) fin >> edge[i] >> _;
    fin.close();

    hipMallocManaged(&L, sizeof(int)*(*NUM_NODES)); my_memset_order(L, 0, *NUM_NODES);
    hipMallocManaged(&R, sizeof(int)*(*NUM_NODES)); my_memset_order(R, 0, *NUM_NODES);
    hipMallocManaged(&P, sizeof(int)*(*NUM_NODES)); my_memset_order(P, 0, *NUM_NODES);
    hipMallocManaged(&Q, sizeof(int)*(*NUM_NODES)); my_memset_order(Q, 0, *NUM_NODES);

    maximal_bic_enum(NUM_NODES, NUM_EDGES, node, edge, L, R, P, Q, ONE);

    hipFree(node);
    hipFree(edge);
    hipFree(NUM_EDGES);
    hipFree(NUM_NODES);
}