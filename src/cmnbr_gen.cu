#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <bits/stdc++.h>
#include <hip/hip_cooperative_groups.h>
#include <sys/ioctl.h>
using namespace std;
using namespace std::chrono;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define NUM_THDS 512

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

struct EdgePair
{
	int row;
	int col;
    EdgePair(int row_, int col_): row(row_), col(col_) {}
};

typedef struct
{
    int num_rows;
    int num_cols;
    unordered_set<int> row;
    unordered_set<int> col;
} Biclique;

typedef struct
{
    int row_end, r;
    int col_end, c;
    int *row;
    int *col;
} Candidate;

random_device rd;
mt19937 en(rd());
uniform_int_distribution<unsigned long long> rand_64;

char t_ms_idx = 0; long long t_ms_start = 0, t_ms_end = 0; vector<long long> t_ms(13, 0);
void mark_time_ms(short new_init_idx) {
    // t_ms_end = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
    t_ms_end = clock();
    if (new_init_idx >= 0) t_ms_idx = new_init_idx;
    else t_ms[t_ms_idx++] += t_ms_end - t_ms_start;
    t_ms_start = t_ms_end;
    return;
}

char choose_op(int num_rows_cand, int num_cols_cand, int num_rows_bic, int num_cols_bic) {
    // cout << num_rows_cand << ' ' << num_cols_cand << ' ' << num_rows_bic << ' ' << num_cols_bic << ' ';
    if ((num_rows_cand == 0 && num_rows_bic == 1) || (num_cols_cand == 0 && num_cols_bic == 1) || (num_rows_cand | num_cols_cand) == 0) return 'x';
    long long prob_rows = 1, prob_cols = 1;
    if (num_rows_bic != 1 || num_cols_bic != 1) {
        prob_rows = (num_rows_cand != 0) * (num_cols_bic - 1);
        prob_cols = (num_cols_cand != 0) * (num_rows_bic - 1) * 2;
    }
    uniform_int_distribution<> rand_cand(0, prob_rows + prob_cols - 1);
    return (rand_cand(en) < prob_rows ? 'r' : 'c');
}

__global__ void CUDA_TRANS_CSPARSE(int *tmp, Node *node_i, int *edge_i, Node *node_o, int *edge_o, long long *NUM_NODES, int *NUM_EDGES)
{
    grid_group grid = this_grid();
    int num_thds  = blockDim.x * gridDim.x;
    // int num_warps = num_thds >> 5;
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id == 0)
        tmp[0] = node_o[0].start = node_o[0].length = 0;
    grid.sync();
    for (int nid = 1 + id; nid < *NUM_NODES; nid += num_thds)
        node_o[nid].length = 0;
    grid.sync();
    for (int eid = id; eid < *NUM_EDGES; eid += num_thds)
        atomicAdd(&(node_o[edge_i[eid]].length), 1);
    grid.sync();
    for (int nid = 1 + id; nid < *NUM_NODES; nid += num_thds)
        node_o[nid].start = node_o[nid - 1].length;
    grid.sync();
    for (int offset = 1; offset < *NUM_NODES; offset <<= 1) {
        for (int nid = *NUM_NODES - (num_thds - id); nid >= offset; nid -= num_thds)
            tmp[nid] = node_o[nid - offset].start + node_o[nid].start;
        grid.sync();
        for (int nid = *NUM_NODES - (num_thds - id); nid >= offset; nid -= num_thds)
            node_o[nid].start = tmp[nid];
        grid.sync();
    }
    // for (int nid = id >> 5; nid < *NUM_NODES; nid += num_warps)
    //     for (int eid = node_i[nid].start + threadIdx.x & 0x1f, eid_end = node_i[nid].start + node_i[nid].length; eid < eid_end; eid += 32)
    //         edge_o[atomicAdd(&(tmp[edge_i[eid]]), 1)] = nid;
    for (int nid = 0; nid < *NUM_NODES; nid++) {
        for (int eid = node_i[nid].start + id, eid_end = node_i[nid].start + node_i[nid].length; eid < eid_end; eid += num_thds)
            edge_o[tmp[edge_i[eid]]++] = nid;
        grid.sync();
    }
}

int main(int argc, char* argv[])
{
    string str_dataset = argv[1];
    // printf("\033[0;1;33m");
    cout << str_dataset.substr(str_dataset.rfind('/')+1) << "\n";
    // printf("\033[0;1m");
	ifstream fin;
    int _, *NUM_EDGES, SOURCE;
    long long *NUM_NODES;
    hipMallocManaged(&NUM_EDGES, sizeof(int));
    hipMallocManaged(&NUM_NODES, sizeof(long long));
    fin.open(argv[1]);
    fin >> *NUM_NODES >> *NUM_EDGES >> SOURCE;

	Node* node;
	int* edge;
    hipMallocManaged(&node, sizeof(Node)*(*NUM_NODES));
    hipMallocManaged(&edge, sizeof(int)*(*NUM_EDGES));
    for(int i=0;i<*NUM_NODES;i++) fin >> node[i].start >> node[i].length;
    for(int i=0;i<*NUM_EDGES;i++) fin >> edge[i] >> _;
    fin.close();

    Node *node_r = node, *node_c;
	int *edge_r = edge, *edge_c, *tmp;
    // Edge_Pair *edge_p;
    hipMallocManaged(&tmp, sizeof(int)*(*NUM_NODES));
    hipMallocManaged(&node_c, sizeof(Node)*(*NUM_NODES));
    hipMallocManaged(&edge_c, sizeof(int)*(*NUM_EDGES));
    // hipMallocManaged(&edge_p, sizeof(Edge_Pair)*(*NUM_EDGES));
    
    cout << "Nodes: " << *NUM_NODES << "\n";
    cout << "Edges: " << *NUM_EDGES << "\n";

    int numBlocksPerSM = 1;
    int numThreads = NUM_THDS;
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_TRANS_CSPARSE, numThreads, 0);
    dim3 num_blocks_TRANS_CSPARSE(deviceProp.multiProcessorCount * numBlocksPerSM, 1, 1);
    dim3 block_size(numThreads, 1, 1);
    void *kernelArgs_TRANS_CSR2CSC[] = {&tmp, &node_r, &edge_r, &node_c, &edge_c, &NUM_NODES, &NUM_EDGES};
    void *kernelArgs_TRANS_CSC2CSR[] = {&tmp, &node_c, &edge_c, &node_r, &edge_r, &NUM_NODES, &NUM_EDGES};
    cout << "block_size: " << numThreads << "\n";
    cout << "num_blocks_TRANS_CSPARSE:   " << num_blocks_TRANS_CSPARSE.x << "\n";
    
    // cout << "Kernel Start\n";
    hipLaunchCooperativeKernel((void*)CUDA_TRANS_CSPARSE, num_blocks_TRANS_CSPARSE, block_size, kernelArgs_TRANS_CSR2CSC);
    hipDeviceSynchronize();
    hipLaunchCooperativeKernel((void*)CUDA_TRANS_CSPARSE, num_blocks_TRANS_CSPARSE, block_size, kernelArgs_TRANS_CSC2CSR);
    hipDeviceSynchronize();
    hipFree(tmp);

    bool check_r = 1, check_c = 1;
    for (int i = 0; i < *NUM_NODES; i++)
        for (int j_ = node_r[i].start, j__end = j_ + node_r[i].length - 1; j_ < j__end; j_++)
            if (edge_r[j_] >= edge_r[j_ + 1]) check_r = 0;
    for (int i = 0; i < *NUM_NODES; i++)
        for (int j_ = node_c[i].start, j__end = j_ + node_c[i].length - 1; j_ < j__end; j_++)
            if (edge_c[j_] >= edge_c[j_ + 1]) check_c = 0;
    cout << "order check: CSR(" << (check_r ? "PASS" : "FAIL") << "), CSC(" << (check_c ? "PASS" : "FAIL") << ")\n";
    
    long long preprocess_time = time(0);
    uniform_int_distribution<> rand_node(0, *NUM_NODES - 1);
    uniform_int_distribution<> rand_edge(0, *NUM_EDGES - 1);
    vector<Biclique*> biclique;
    vector<EdgePair*> edge_lonely;
    Biclique *bic_ptr;
    Candidate *cand_ptr;
    int biclique_cover = 0, biclique_score = 0;
    long long *NUM_BICS;
    int *NUM_BIC_EDGES;
    hipMallocManaged(&NUM_BICS, sizeof(long long));
    hipMallocManaged(&NUM_BIC_EDGES, sizeof(int));
    *NUM_BICS = *NUM_BIC_EDGES = 0;
    // progress bar
    short bar_size_max = 0, set_width = log10(*NUM_NODES - 1) + 1;
    string str_bar[8] = {"", "▏", "▎", "▍", "▌", "▋", "▊", "▉"};
    
    for (int nid_max_r = *NUM_NODES - 1, nid_now_r = 0, eid_now_r = 0, cid, rid; true; ) {

        mark_time_ms(0);
        
        rid = nid_now_r;
        cid = edge_r[eid_now_r];
        // cout << "\nFrom edge " << eid_now_r << ": row " << (rid) << ", col " << (cid) << "\n";

        cand_ptr = new Candidate;
        bic_ptr = new Biclique;
        cand_ptr->r = cand_ptr->c = 0;
        bic_ptr->row.insert(rid); bic_ptr->col.insert(cid);
        bic_ptr->num_rows = bic_ptr->num_cols = 1;

        mark_time_ms(-1);

        cand_ptr->col = new int[(cand_ptr->col_end = node_r[rid].length)--];
        copy(edge_r + node_r[rid].start, edge_r + node_r[rid].start + node_r[rid].length, cand_ptr->col);
        
        cand_ptr->row = new int[(cand_ptr->row_end = node_c[cid].length)--];
        copy(edge_c + node_c[cid].start, edge_c + node_c[cid].start + node_c[cid].length, cand_ptr->row);

        mark_time_ms(-1);

        for (int cols = cand_ptr->col_end; cols > 0; ) {
            swap(cand_ptr->col[rand_64(en)%cols], cand_ptr->col[cols - 1]);
            if (cand_ptr->col[--cols] == cid)
                cand_ptr->col[cols] = cand_ptr->col[cand_ptr->col_end];
        }
        for (int rows = cand_ptr->row_end; rows > 0; ) {
            swap(cand_ptr->row[rand_64(en)%rows], cand_ptr->row[rows - 1]);
            if (cand_ptr->row[--rows] == rid)
                cand_ptr->row[rows] = cand_ptr->row[cand_ptr->row_end];
        }

        mark_time_ms(-1);
        
        for (char op; true; ) {
            op = choose_op(cand_ptr->row_end - cand_ptr->r, cand_ptr->col_end - cand_ptr->c, bic_ptr->num_rows, bic_ptr->num_cols);
            // cout << "Operation " << op << "\n";
            if (op == 'r') {
                rid = cand_ptr->row[cand_ptr->r++];
                unordered_set<int> col_tmp(edge_r + node_r[rid].start, edge_r + node_r[rid].start + node_r[rid].length);
                for (const auto &col : bic_ptr->col) col_tmp.erase(col);
                // cout << "row " << rid << " successed.\n";
                bic_ptr->row.insert(rid);
                bic_ptr->num_rows++;
                for (int c = cand_ptr->c; c < cand_ptr->col_end; )
                    if (col_tmp.erase(cand_ptr->col[c])) c++;
                    else cand_ptr->col[c] = cand_ptr->col[--cand_ptr->col_end];
            }
            else if (op == 'c') {
                cid = cand_ptr->col[cand_ptr->c++];
                unordered_set<int> row_tmp(edge_c + node_c[cid].start, edge_c + node_c[cid].start + node_c[cid].length);
                for (const auto &row : bic_ptr->row) row_tmp.erase(row);
                // cout << "col " << cid << " successed.\n";
                bic_ptr->col.insert(cid);
                bic_ptr->num_cols++;
                for (int r = cand_ptr->r; r < cand_ptr->row_end; )
                    if (row_tmp.erase(cand_ptr->row[r])) r++;
                    else cand_ptr->row[r] = cand_ptr->row[--cand_ptr->row_end];
            }
            else break;
        }

        mark_time_ms(-1);

        delete [] cand_ptr->row;
        delete [] cand_ptr->col;
        delete cand_ptr;

        mark_time_ms(-1);

        // if ((bic_ptr->num_rows == 1) ^ (bic_ptr->num_cols == 1)) eid_now_r++;
        if ((bic_ptr->num_rows - 1) * (bic_ptr->num_cols - 1) < 8 && ((bic_ptr->num_rows != 1) || (bic_ptr->num_cols != 1))) eid_now_r++;
        else {
            // cout << "Delete\n";
            for (const auto &row : bic_ptr->row) {
                int eid = node_r[row].start, eid_end = eid + node_r[row].length;
                if (row == nid_now_r)
                    while (eid < eid_end)
                        if (bic_ptr->col.find(edge_r[eid]) == bic_ptr->col.end()) eid++;
                        else if (eid < eid_now_r) {
                            swap(edge_r[eid], edge_r[--eid_now_r]);
                            edge_r[eid_now_r] = edge_r[--eid_end];
                        }
                        else edge_r[eid] = edge_r[--eid_end];
                else
                    while (eid < eid_end)
                        if (bic_ptr->col.find(edge_r[eid]) == bic_ptr->col.end()) eid++;
                        else edge_r[eid] = edge_r[--eid_end];
                node_r[row].length = eid_end - node_r[row].start;
            }
            for (const auto &col : bic_ptr->col) {
                int eid = node_c[col].start, eid_end = eid + node_c[col].length;
                while (eid < eid_end)
                    if (bic_ptr->row.find(edge_c[eid]) == bic_ptr->row.end()) eid++;
                    else edge_c[eid] = edge_c[--eid_end];
                node_c[col].length = eid_end - node_c[col].start;
            }
            if (bic_ptr->num_rows == 1 && bic_ptr->num_cols == 1) {
                edge_lonely.push_back(new EdgePair(*begin(bic_ptr->row), *begin(bic_ptr->col)));
                delete bic_ptr;
            }
            else {
                (*NUM_BICS)++;
                *NUM_BIC_EDGES += bic_ptr->num_cols;
                biclique_cover += bic_ptr->num_rows * bic_ptr->num_cols;
                biclique_score += (bic_ptr->num_rows - 1) * (bic_ptr->num_cols - 1);
                biclique.push_back(bic_ptr);
            }
        }

        mark_time_ms(-1);

        while (node_r[nid_max_r].length == 0) nid_max_r--;
        if (eid_now_r >= node_r[nid_max_r].start + node_r[nid_max_r].length) break;
        if (eid_now_r == node_r[nid_now_r].start + node_r[nid_now_r].length) {
            while (node_r[++nid_now_r].length == 0) ;
            eid_now_r = node_r[nid_now_r].start;
        }

        mark_time_ms(-1);

        // // progress bar
        // struct winsize w;
        // ioctl(STDOUT_FILENO, TIOCGWINSZ, &w);
        // short bar_width = (w.ws_col - (short)(6 + set_width * 2)) << 3;
        // if (bar_width < 8) continue;
        // long long bar_size = nid_now_r * bar_width / nid_max_r;
        // if (bar_size > bar_size_max) {
        //     bar_size_max = bar_size;
        //     // cout << (rand()%((bar_size>>3)+1) ? "." : ".\n");
        //     cout << "\33[1A|";
        //     for (short i = bar_size >> 3; i > 0; i--) cout << "█";
        //     cout << str_bar[bar_size % 8];
        //     for (short i = bar_width - bar_size >> 3; i > 0; i--) cout << " ";
        //     cout << "| " << setw(set_width) << nid_now_r << "/" << nid_max_r << " |\n";
        // }

        mark_time_ms(-1);
    }

    mark_time_ms(8);

    for (int rid, cid; !edge_lonely.empty(); edge_lonely.pop_back()) {
        EdgePair* edge_ptr = edge_lonely.back();
        rid = edge_ptr->row;
        cid = edge_ptr->col;
        delete edge_ptr;
        edge_r[node_r[rid].start + (node_r[rid].length++)] = cid;
        // edge_c[node_c[cid].start + (node_c[cid].length++)] = rid;
    }

    mark_time_ms(-1);

    hipFree(node_c);
    hipFree(edge_c);
    hipMallocManaged(&node_c, sizeof(Node)*(*NUM_BICS));     // biclique list (node)
    hipMallocManaged(&edge_c, sizeof(int)*(*NUM_BIC_EDGES)); // biclique list (edge)

    mark_time_ms(-1);

    for (int bid = 0, eid = 0; bid < *NUM_BICS; bid++) {
        bic_ptr = biclique[bid];
        for (const auto &row : bic_ptr->row)
            edge_r[node_r[row].start + (node_r[row].length++)] = ~bid;
        node_c[bid].start  = eid;
        node_c[bid].length = bic_ptr->num_cols;
        for (const auto &col : bic_ptr->col)
            edge_c[eid++] = col;
        delete bic_ptr;
    }
    biclique.clear();

    mark_time_ms(-1);

    *NUM_EDGES = node[0].length;
    for (int nid = 1; nid < *NUM_NODES; nid++) {
        int eid_start = node[nid].start;
        node[nid].start = *NUM_EDGES;
        for (int eid = eid_start, eid_end = eid + node[nid].length; eid < eid_end; eid++) {
            edge[(*NUM_EDGES)++] = edge[eid];
        }
    }

    mark_time_ms(-1);

    ofstream fout;
    fout.open(argv[2]);
    fout << *NUM_NODES << ' ' << *NUM_EDGES << "\n\n";
    for (int i = 0; i < *NUM_NODES; i++)
        fout << node[i].start << ' ' << node[i].length << "\n";
    fout << "\n";
    for (int i = 0; i < *NUM_EDGES; i++)
        fout << edge[i] << "\n";
    fout << "\n";
    fout << *NUM_BICS << ' ' << *NUM_BIC_EDGES << "\n\n";
    for (int i = 0; i < *NUM_BICS; i++)
        fout << node_c[i].start << ' ' << node_c[i].length << "\n";
    fout << "\n";
    for (int i = 0; i < *NUM_BIC_EDGES; i++)
        fout << edge_c[i] << "\n";
    fout << "\n";
    fout.close();

    cout << "t_ms:"; for (short i = 0; i < t_ms.size(); i++) cout << ' ' << (t_ms[i] >> 10); cout << "\n";
    cout << "Process time is " << (preprocess_time = time(0) - preprocess_time) << "s\n";
    cout << "found " << *NUM_BICS << " bicliques" << "\n";
    cout << "biclique cover: " << biclique_cover << "\n";
    cout << "avg cover: " << (double)biclique_cover / (*NUM_BICS) << "\n";
    cout << "biclique score: " << biclique_score << "\n";
    cout << "avg score: " << (double)biclique_score / (*NUM_BICS) << "\n";

    hipFree(NUM_EDGES);
    hipFree(NUM_NODES);
    hipFree(NUM_BICS);
    hipFree(NUM_BIC_EDGES);
    hipFree(node_r);
    hipFree(edge_r);
    hipFree(node_c);
    hipFree(edge_c);
}