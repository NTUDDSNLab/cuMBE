#include <src/header.cuh>
#include <src/transpose.cuh>
#include <src/mbe_cpu.cuh>
#include <src/mbe_cpu_lp.cuh>
#include <src/mbe_gpu_1b.cuh>
#include <src/mbe_gpu.cuh>

void maximal_bic_enum_MineLMBC(int *NUM_L, int *NUM_R, int *NUM_EDGES, Node *node_r, int *edge_r,
                               unordered_set<int> X, unordered_set<int> gammaX, set<int> tailX) {
    unordered_set<int> gammaXprime;
    for (const auto &v: tailX) {
        gammaXprime.clear();
        for (int eid = node_r[v].start, eid_end = eid + node_r[v].length; eid < eid_end; eid++) {
            int u = edge_r[eid];
            if (gammaX.find(u) != gammaX.end())
                gammaXprime.insert(u);
        }
        if (gammaXprime.size() < MIN_SH)
            tailX.erase(v);
    }
    if (X.size() + tailX.size() < MIN_SH)
        return;
    for (const auto &v: tailX) {
        tailX.erase(v);
        unordered_set<int> Y = X;
        Y.insert(v);
        if (X.size() + tailX.size() + 1 > MIN_SH) {
            for (const auto &v_: tailX) {
                int num_N_v = 0;
                for (int eid = node_r[v_].start, eid_end = eid + node_r[v_].length; eid < eid_end; eid++) {
                    int u = edge_r[eid];
                    if (gammaXprime.find(u) != gammaXprime.end())
                        ++num_N_v;
                }
                if (num_N_v == gammaXprime.size())
                    Y.insert(v_);
            }
            
        }
    }
}

int main(int argc, char* argv[])
{
    time_t tmNow = time(0);
    string dataset = argv[1];
    dataset = dataset.substr(dataset.rfind('/')+1);

    Node *node_l, *node_r;
	int *edge_l, *edge_r, *tmp;
    int *NUM_L, *NUM_R, *NUM_EDGES, _;
    int *num_mb, *time_section;
    // MBE
    int *u2L, *v2P, *v2Q, *L, *R, *P, *Q;
    int *x, *L_lp, *R_lp, *P_lp, *Q_lp;
    int *Q_rm, *L_buf, *num_N_u, *pre_min;
    // MBE_82
    int *g_u2L, *g_v2P, *g_v2Q, *g_L, *g_R, *g_P, *g_Q;
    int *g_x, *g_L_lp, *g_R_lp, *g_P_lp, *g_Q_lp;
    int *g_Q_rm, *g_L_buf, *g_num_N_u, *g_pre_min;
    int *ori_P;
    int *ori_P1, *ori_Q1, *ori_L1, *P_ptr1, *fix_P_ptr1;
    hipMallocManaged(&NUM_EDGES   , sizeof(int));
    hipMallocManaged(&NUM_L       , sizeof(int));
    hipMallocManaged(&NUM_R       , sizeof(int));
    hipMallocManaged(&num_mb      , sizeof(int));
    hipMallocManaged(&time_section, sizeof(int)*NUM_CLK);
    *num_mb = 0;
    my_memset(time_section, 0, NUM_CLK);

    ifstream fin;
    fin.open(argv[1]);
    fin >> *NUM_R >> *NUM_L >> *NUM_EDGES;
    hipMallocManaged(&tmp   , sizeof(int )*(*NUM_L    ));
    hipMallocManaged(&node_l, sizeof(Node)*(*NUM_L    ));
    hipMallocManaged(&edge_l, sizeof(int )*(*NUM_EDGES));
    hipMallocManaged(&node_r, sizeof(Node)*(*NUM_R    ));
    hipMallocManaged(&edge_r, sizeof(int )*(*NUM_EDGES));
    for (int i = 0; i < *NUM_R    ; i++) fin >> node_r[i].start >> node_r[i].length;
    for (int i = 0; i < *NUM_EDGES; i++) fin >> edge_r[i] >> _;
    fin.close();

    int numBlocksPerSM;
    int numThreads = NUM_THDS;
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_MBE_82, numThreads, 0);
    int numBlocks_max = deviceProp.multiProcessorCount * numBlocksPerSM;
    int numBlocks = NUM_BLKS > numBlocks_max ? numBlocks_max : \
                    NUM_BLKS == 0 ? 1 : NUM_BLKS < 0 ? 0 : NUM_BLKS;
    dim3 num_blocks_TRANSPOSE(numBlocks, 1, 1);
    dim3 num_blocks_MBE(1, 1, 1);
    dim3 num_blocks_MBE_82(numBlocks, 1, 1);
    dim3 block_size(numThreads, 1, 1);

    bool swap_RL = *NUM_R > *NUM_L;
    if (swap_RL) {
        swap(*NUM_L, *NUM_R);
        swap(node_l, node_r);
        swap(edge_l, edge_r);
    }

    // MBE
    hipMallocManaged(&u2L    , sizeof(int)*(*NUM_L)); my_memset_order(u2L, 0, *NUM_L);
    hipMallocManaged(&v2P    , sizeof(int)*(*NUM_R)); my_memset_order(v2P, 0, *NUM_R);
    hipMallocManaged(&v2Q    , sizeof(int)*(*NUM_R)); my_memset_order(v2Q, 0, *NUM_R);
    hipMallocManaged(&L      , sizeof(int)*(*NUM_L)); my_memset_order(L  , 0, *NUM_L);
    hipMallocManaged(&R      , sizeof(int)*(*NUM_R)); my_memset_order(R  , 0, *NUM_R);
    hipMallocManaged(&P      , sizeof(int)*(*NUM_R)); my_memset_order(P  , 0, *NUM_R);
    hipMallocManaged(&Q      , sizeof(int)*(*NUM_R)); my_memset_order(Q  , 0, *NUM_R);
    hipMallocManaged(&x      , sizeof(int)*(*NUM_R)); my_memset(x   ,     -1, *NUM_R);
    hipMallocManaged(&L_lp   , sizeof(int)*(*NUM_R)); my_memset(L_lp, *NUM_L, *NUM_R);
    hipMallocManaged(&R_lp   , sizeof(int)*(*NUM_R)); my_memset(R_lp,      0, *NUM_R);
    hipMallocManaged(&P_lp   , sizeof(int)*(*NUM_R)); my_memset(P_lp, *NUM_R, *NUM_R);
    hipMallocManaged(&Q_lp   , sizeof(int)*(*NUM_R)); my_memset(Q_lp,      0, *NUM_R);
    hipMallocManaged(&Q_rm   , sizeof(int)*(*NUM_R)); my_memset(Q_rm,    INF, *NUM_R);
    hipMallocManaged(&L_buf  , sizeof(int)*(*NUM_L)); my_memset(L_buf,     0, *NUM_L);
    hipMallocManaged(&num_N_u, sizeof(int)*(*NUM_R)); my_memset(num_N_u,   0, *NUM_R);
    hipMallocManaged(&pre_min, sizeof(int)*(*NUM_R)); my_memset(pre_min,   1, *NUM_R);
    // MBE_82
    hipMallocManaged(&g_u2L    , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )     g_u2L[i] =     u2L[i % (*NUM_L)];
    hipMallocManaged(&g_v2P    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_v2P[i] =     v2P[i % (*NUM_R)];
    hipMallocManaged(&g_v2Q    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_v2Q[i] =     v2Q[i % (*NUM_R)];
    hipMallocManaged(&g_L      , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )       g_L[i] =       L[i % (*NUM_L)];
    hipMallocManaged(&g_R      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_R[i] =       R[i % (*NUM_R)];
    hipMallocManaged(&g_P      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_P[i] =       P[i % (*NUM_R)];
    hipMallocManaged(&g_Q      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_Q[i] =       Q[i % (*NUM_R)];
    hipMallocManaged(&g_x      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_x[i] =       x[i % (*NUM_R)];
    hipMallocManaged(&g_L_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_L_lp[i] =    L_lp[i % (*NUM_R)];
    hipMallocManaged(&g_R_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_R_lp[i] =    R_lp[i % (*NUM_R)];
    hipMallocManaged(&g_P_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_P_lp[i] =    P_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_Q_lp[i] =    Q_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_rm   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_Q_rm[i] =    Q_rm[i % (*NUM_R)];
    hipMallocManaged(&g_L_buf  , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )   g_L_buf[i] =   L_buf[i % (*NUM_L)];
    hipMallocManaged(&g_num_N_u, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_num_N_u[i] = num_N_u[i % (*NUM_R)];
    hipMallocManaged(&g_pre_min, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_pre_min[i] = pre_min[i % (*NUM_R)];
    hipMallocManaged(&ori_P     , sizeof(int)*(*NUM_R));
    hipMallocManaged(&ori_P1    , sizeof(int)*(*NUM_R)*numBlocks);
    hipMallocManaged(&ori_Q1    , sizeof(int)         *numBlocks);
    hipMallocManaged(&ori_L1    , sizeof(int)*(*NUM_L)*numBlocks);
    hipMallocManaged(&P_ptr1    , sizeof(int)         *numBlocks);
    hipMallocManaged(&fix_P_ptr1, sizeof(int)         *numBlocks);

    void *kernelArgs_CSR2CSC[] = {&tmp, &node_r, &edge_r, &node_l, &edge_l, &NUM_R, &NUM_L, &NUM_EDGES};
    void *kernelArgs_CSC2CSR[] = {&tmp, &node_l, &edge_l, &node_r, &edge_r, &NUM_L, &NUM_R, &NUM_EDGES};
    void *kernelArgs_MBE[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node_r, &edge_r, &u2L, &L, &R, &P, &Q, &x, &L_lp, &R_lp, &P_lp, &Q_lp};
    void *kernelArgs_MBE_82[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node_l, &edge_l, &node_r, &edge_r,
                                 &g_u2L, &g_v2P, &g_v2Q, &g_L, &g_R, &g_P, &g_Q, &g_Q_rm, &g_x, &g_L_lp, &g_R_lp, &g_P_lp, &g_Q_lp, &g_L_buf, &g_num_N_u, &g_pre_min,
                                 &ori_P, &ori_P1, &ori_Q1, &ori_L1, &P_ptr1, &fix_P_ptr1, &num_mb, &time_section};

    string algo;
    switch (NUM_BLKS) {
        case -2: algo = "CPU"   ; break;
        case -1: algo = "CPU_lp"; break;
        case  0: algo = "GPU_1B"; break;
        default: algo = "GPU"   ; break;
    }
    string filename = dataset.substr(0, dataset.rfind('.'));
    filename += "_";
    filename += algo;
    cout << filename << endl;
    
    ofstream fout;
    fout.open("result/"+filename);

#ifdef DEBUG
    if (algo == "GPU") {
        cout << "\33[2J\33[1;1H";
    }
#endif /* DEBUG */
    cout << "date/time: "<< ctime(&tmNow);
    cout << "algorithm: " << algo << endl;
    cout << "dataset: " << dataset << endl;
    cout << "|R|: " << *NUM_R << ", |L|: " << *NUM_L << ", |E|: " << *NUM_EDGES << endl;
    cout << "grid_size: " << numBlocks << ", block_size: " << numThreads << endl;

    fout << "date/time: "<< ctime(&tmNow);
    fout << "algorithm: " << algo << endl;
    fout << "dataset: " << dataset << endl;
    fout << "|R|: " << *NUM_R << ", |L|: " << *NUM_L << ", |E|: " << *NUM_EDGES << endl;
    fout << "grid_size: " << numBlocks << ", block_size: " << numThreads << endl;

    hipLaunchCooperativeKernel((void*)CUDA_TRANSPOSE, num_blocks_TRANSPOSE, block_size, swap_RL ? kernelArgs_CSC2CSR : kernelArgs_CSR2CSC);
    hipDeviceSynchronize();
    my_memset_sort(ori_P, 0, *NUM_R, node_r);
    
    hipMemPrefetchAsync(&NUM_EDGES   , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&NUM_L       , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&NUM_R       , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&num_mb      , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&time_section, sizeof(int)*NUM_CLK, device, NULL);

    hipMemPrefetchAsync(&node_l, sizeof(Node)*(*NUM_L    ), device, NULL);
    hipMemPrefetchAsync(&edge_l, sizeof(int )*(*NUM_EDGES), device, NULL);
    hipMemPrefetchAsync(&node_r, sizeof(Node)*(*NUM_R    ), device, NULL);
    hipMemPrefetchAsync(&edge_r, sizeof(int )*(*NUM_EDGES), device, NULL);
    
    hipMemPrefetchAsync(g_u2L    , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_v2P    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_v2Q    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_L      , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_R      , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_P      , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_Q      , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_x      , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_L_lp   , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_R_lp   , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_P_lp   , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_Q_lp   , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_Q_rm   , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_L_buf  , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_num_N_u, sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_pre_min, sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(ori_P        , sizeof(int)*(*NUM_R)          , device, NULL);
    hipMemPrefetchAsync(ori_P1       , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(ori_Q1       , sizeof(int)         *numBlocks, device, NULL);
    hipMemPrefetchAsync(ori_L1       , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(P_ptr1       , sizeof(int)         *numBlocks, device, NULL);
    hipMemPrefetchAsync(fix_P_ptr1   , sizeof(int)         *numBlocks, device, NULL);

    hipDeviceSynchronize();

    int stat;
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if (algo == "CPU")
        maximal_bic_enum_set(NUM_L, NUM_R, NUM_EDGES, node_r, edge_r, u2L, L, R, P, Q, x, L_lp, R_lp, P_lp, Q_lp);
    else if (algo == "CPU_lp")
        maximal_bic_enum(NUM_L, NUM_R, NUM_EDGES, node_r, edge_r, u2L, L, R, P, Q, x, L_lp, R_lp, P_lp, Q_lp);
    else if (algo == "GPU_1B")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE, num_blocks_MBE, block_size, kernelArgs_MBE);
    else if (algo == "GPU")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_82, num_blocks_MBE_82, block_size, kernelArgs_MBE_82);
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // cout << "status: " << stat << endl;
    cout << "maximal bicliques: " << *num_mb << endl;
    cout << "time:";
    for (int i = 0; i < NUM_CLK; i++)
        cout << " " << time_section[i];
    cout << endl;
    cout << "runtime (s): " << time/1000 << endl;

    fout << "maximal bicliques: " << *num_mb << endl;
    fout << "time:";
    for (int i = 0; i < NUM_CLK; i++)
        fout << " " << time_section[i];
    fout << endl;
    fout << "runtime (s): " << time/1000 << endl;
#ifdef DEBUG
    if (algo == "GPU")
        cout << "\33[" << (numBlocks-1) / WORDS_1ROW + 10 << ";1H";
#endif /* DEBUG */

    hipFree(tmp);
    hipFree(node_l);
    hipFree(edge_l);
    hipFree(node_r);
    hipFree(edge_r);
    hipFree(NUM_L);
    hipFree(NUM_R);
    hipFree(NUM_EDGES);
    hipFree(num_mb);
    hipFree(time_section);
    // MBE
    hipFree(u2L);
    hipFree(v2P);
    hipFree(v2Q);
    hipFree(L);
    hipFree(R);
    hipFree(P);
    hipFree(Q);
    hipFree(x);
    hipFree(L_lp);
    hipFree(R_lp);
    hipFree(P_lp);
    hipFree(Q_lp);
    hipFree(L_buf);
    hipFree(num_N_u);
    hipFree(pre_min);
    // MBE_82
    hipFree(g_u2L);
    hipFree(g_v2P);
    hipFree(g_v2Q);
    hipFree(g_L);
    hipFree(g_R);
    hipFree(g_P);
    hipFree(g_Q);
    hipFree(g_x);
    hipFree(g_L_lp);
    hipFree(g_R_lp);
    hipFree(g_P_lp);
    hipFree(g_Q_lp);
    hipFree(g_L_buf);
    hipFree(g_num_N_u);
    hipFree(g_pre_min);
    hipFree(ori_P);
    hipFree(ori_P1);
    hipFree(ori_Q1);
    hipFree(ori_L1);
    hipFree(P_ptr1);
    hipFree(fix_P_ptr1);
}