#include <src/header.cuh>
#include <src/transpose.cuh>
#include <src/mbe_cpu.cuh>
#include <src/mbe_cpu_lp.cuh>
#include <src/mbe_gpu_1b.cuh>
#include <src/mbe_gpu.cuh>

void maximal_bic_enum_MineLMBC(int *NUM_L, int *NUM_R, int *NUM_EDGES, Node *node_r, int *edge_r,
                               unordered_set<int> X, unordered_set<int> gammaX, set<int> tailX) {
    unordered_set<int> gammaXprime;
    for (const auto &v: tailX) {
        gammaXprime.clear();
        for (int eid = node_r[v].start, eid_end = eid + node_r[v].length; eid < eid_end; eid++) {
            int u = edge_r[eid];
            if (gammaX.find(u) != gammaX.end())
                gammaXprime.insert(u);
        }
        if (gammaXprime.size() < MIN_SH)
            tailX.erase(v);
    }
    if (X.size() + tailX.size() < MIN_SH)
        return;
    for (const auto &v: tailX) {
        tailX.erase(v);
        unordered_set<int> Y = X;
        Y.insert(v);
        if (X.size() + tailX.size() + 1 > MIN_SH) {
            for (const auto &v_: tailX) {
                int num_N_v = 0;
                for (int eid = node_r[v_].start, eid_end = eid + node_r[v_].length; eid < eid_end; eid++) {
                    int u = edge_r[eid];
                    if (gammaXprime.find(u) != gammaXprime.end())
                        ++num_N_v;
                }
                if (num_N_v == gammaXprime.size())
                    Y.insert(v_);
            }
            
        }
    }
}

int main(int argc, char* argv[])
{
    time_t tmNow = time(0);
    string dataset = argv[1];
    dataset = dataset.substr(dataset.rfind('/')+1);

    Node *node_l, *node_r;
	int *edge_l, *edge_r, *tmp;
    int *NUM_L, *NUM_R, *NUM_EDGES, _;
    // MBE
    int *u2L, *L, *R, *P, *Q;
    int *x, *L_lp, *R_lp, *P_lp, *Q_lp;
    int *Q_rm, *L_buf;
    // MBE_82
    int *g_u2L, *g_L, *g_R, *g_P, *g_Q;
    int *g_x, *g_L_lp, *g_R_lp, *g_P_lp, *g_Q_lp;
    int *g_Q_rm, *g_L_buf, *ori_P;
    hipMallocManaged(&NUM_EDGES, sizeof(int));
    hipMallocManaged(&NUM_L    , sizeof(int));
    hipMallocManaged(&NUM_R    , sizeof(int));

    ifstream fin;
    fin.open(argv[1]);
    fin >> *NUM_R >> *NUM_L >> *NUM_EDGES;
    hipMallocManaged(&tmp   , sizeof(int )*(*NUM_L    ));
    hipMallocManaged(&node_l, sizeof(Node)*(*NUM_L    ));
    hipMallocManaged(&edge_l, sizeof(int )*(*NUM_EDGES));
    hipMallocManaged(&node_r, sizeof(Node)*(*NUM_R    ));
    hipMallocManaged(&edge_r, sizeof(int )*(*NUM_EDGES));
    for (int i = 0; i < *NUM_R    ; i++) fin >> node_r[i].start >> node_r[i].length;
    for (int i = 0; i < *NUM_EDGES; i++) fin >> edge_r[i] >> _;
    fin.close();

    int numBlocksPerSM;
    int numThreads = NUM_THDS;
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_MBE_82, numThreads, 0);
    int numBlocks_max = deviceProp.multiProcessorCount * numBlocksPerSM;
    int numBlocks = NUM_BLKS > numBlocks_max ? numBlocks_max : \
                    NUM_BLKS == 0 ? 1 : NUM_BLKS < 0 ? 0 : NUM_BLKS;
    dim3 num_blocks_TRANSPOSE(numBlocks, 1, 1);
    dim3 num_blocks_MBE(1, 1, 1);
    dim3 num_blocks_MBE_82(numBlocks, 1, 1);
    dim3 block_size(numThreads, 1, 1);

    bool swap_RL = *NUM_R > *NUM_L;
    if (swap_RL) {
        // cout << "RETURN\n"; return 0;
        swap(*NUM_L, *NUM_R);
        swap(node_l, node_r);
        swap(edge_l, edge_r);
    }

    // MBE
    hipMallocManaged(&u2L  , sizeof(int)*(*NUM_L)); my_memset_order(u2L, 0, *NUM_L);
    hipMallocManaged(&L    , sizeof(int)*(*NUM_L)); my_memset_order(L  , 0, *NUM_L);
    hipMallocManaged(&R    , sizeof(int)*(*NUM_R)); my_memset_order(R  , 0, *NUM_R);
    hipMallocManaged(&P    , sizeof(int)*(*NUM_R)); my_memset_order(P  , 0, *NUM_R);
    hipMallocManaged(&Q    , sizeof(int)*(*NUM_R)); my_memset_order(Q  , 0, *NUM_R);
    hipMallocManaged(&x    , sizeof(int)*(*NUM_R)); my_memset(x   ,     -1, *NUM_R);
    hipMallocManaged(&L_lp , sizeof(int)*(*NUM_R)); my_memset(L_lp, *NUM_L, *NUM_R);
    hipMallocManaged(&R_lp , sizeof(int)*(*NUM_R)); my_memset(R_lp,      0, *NUM_R);
    hipMallocManaged(&P_lp , sizeof(int)*(*NUM_R)); my_memset(P_lp, *NUM_R, *NUM_R);
    hipMallocManaged(&Q_lp , sizeof(int)*(*NUM_R)); my_memset(Q_lp,      0, *NUM_R);
    hipMallocManaged(&Q_rm , sizeof(int)*(*NUM_R)); my_memset(Q_rm,    INF, *NUM_R);
    hipMallocManaged(&L_buf, sizeof(int)*(*NUM_L)); my_memset(L_buf,     0, *NUM_L);
    // MBE_82
    hipMallocManaged(&g_u2L  , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )   g_u2L[i] =   u2L[i % (*NUM_L)];
    hipMallocManaged(&g_L    , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )     g_L[i] =     L[i % (*NUM_L)];
    hipMallocManaged(&g_R    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_R[i] =     R[i % (*NUM_R)];
    hipMallocManaged(&g_P    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_P[i] =     P[i % (*NUM_R)];
    hipMallocManaged(&g_Q    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_Q[i] =     Q[i % (*NUM_R)];
    hipMallocManaged(&g_x    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_x[i] =     x[i % (*NUM_R)];
    hipMallocManaged(&g_L_lp , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )  g_L_lp[i] =  L_lp[i % (*NUM_R)];
    hipMallocManaged(&g_R_lp , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )  g_R_lp[i] =  R_lp[i % (*NUM_R)];
    hipMallocManaged(&g_P_lp , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )  g_P_lp[i] =  P_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_lp , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )  g_Q_lp[i] =  Q_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_rm , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )  g_Q_rm[i] =  Q_rm[i % (*NUM_R)];
    hipMallocManaged(&g_L_buf, sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; ) g_L_buf[i] = L_buf[i % (*NUM_L)];
    hipMallocManaged(&ori_P, sizeof(int)*(*NUM_R));

    void *kernelArgs_CSR2CSC[] = {&tmp, &node_r, &edge_r, &node_l, &edge_l, &NUM_R, &NUM_L, &NUM_EDGES};
    void *kernelArgs_CSC2CSR[] = {&tmp, &node_l, &edge_l, &node_r, &edge_r, &NUM_L, &NUM_R, &NUM_EDGES};
    void *kernelArgs_MBE[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node_r, &edge_r, &u2L, &L, &R, &P, &Q, &x, &L_lp, &R_lp, &P_lp, &Q_lp};
    void *kernelArgs_MBE_82[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node_l, &edge_l, &node_r, &edge_r, &g_u2L, &g_L, &g_R, &g_P, &g_Q, &g_Q_rm, &g_x, &g_L_lp, &g_R_lp, &g_P_lp, &g_Q_lp, &g_L_buf, &ori_P};

    string algo;
    switch (NUM_BLKS) {
        case -2: algo = "CPU"   ; break;
        case -1: algo = "CPU_lp"; break;
        case  0: algo = "GPU_1B"; break;
        default: algo = "GPU"   ; break;
    }

#ifdef DEBUG
    if (algo == "GPU") {
        cout << "\33[2J\33[1;1H";
    }
#endif /* DEBUG */
    cout << "date/time: "<< ctime(&tmNow);
    cout << "algorithm: " << algo << "\n";
    cout << "dataset: " << dataset << "\n";
    cout << "|R|: " << *NUM_R << ", |L|: " << *NUM_L << ", |E|: " << *NUM_EDGES << "\n";
    cout << "grid_size: " << numBlocks << ", block_size: " << numThreads << "\n";

    // hipLaunchCooperativeKernel((void*)CUDA_MBE_82, num_blocks_MBE_82, block_size, kernelArgs_MBE_82);
    hipLaunchCooperativeKernel((void*)CUDA_TRANSPOSE, num_blocks_TRANSPOSE, block_size, swap_RL ? kernelArgs_CSC2CSR : kernelArgs_CSR2CSC);
    hipDeviceSynchronize();

    my_memset_sort(ori_P, 0, *NUM_R, node_r);
    
    int stat;
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if (algo == "CPU")
        maximal_bic_enum_set(NUM_L, NUM_R, NUM_EDGES, node_r, edge_r, u2L, L, R, P, Q, x, L_lp, R_lp, P_lp, Q_lp);
    else if (algo == "CPU_lp")
        maximal_bic_enum(NUM_L, NUM_R, NUM_EDGES, node_r, edge_r, u2L, L, R, P, Q, x, L_lp, R_lp, P_lp, Q_lp);
    else if (algo == "GPU_1B")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE, num_blocks_MBE, block_size, kernelArgs_MBE);
    else if (algo == "GPU")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_82, num_blocks_MBE_82, block_size, kernelArgs_MBE_82);
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // cout << "status: " << stat << ", ";
    cout << "runtime (s): " << time/1000 << "\n";

    hipFree(tmp);
    hipFree(node_l);
    hipFree(edge_l);
    hipFree(node_r);
    hipFree(edge_r);
    hipFree(NUM_L);
    hipFree(NUM_R);
    hipFree(NUM_EDGES);
    // MBE
    hipFree(u2L);
    hipFree(L);
    hipFree(R);
    hipFree(P);
    hipFree(Q);
    hipFree(x);
    hipFree(L_lp);
    hipFree(R_lp);
    hipFree(P_lp);
    hipFree(Q_lp);
    hipFree(L_buf);
    // MBE_82
    hipFree(g_u2L);
    hipFree(g_L);
    hipFree(g_R);
    hipFree(g_P);
    hipFree(g_Q);
    hipFree(g_x);
    hipFree(g_L_lp);
    hipFree(g_R_lp);
    hipFree(g_P_lp);
    hipFree(g_Q_lp);
    hipFree(g_L_buf);
    hipFree(ori_P);
}