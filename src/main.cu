#include <src/header.cuh>
#include <src/transpose.cuh>
#include <src/mbe_cuMBE.cuh>
#include <src/mbe_noES.cuh>
#include <src/mbe_noRS.cuh>
#include <src/mbe_noWS.cuh>

int main(int argc, char* argv[])
{
    time_t tmNow = time(0);
    string dataset = argv[1];
    dataset = dataset.substr(dataset.rfind('/')+1);

    Node *node_l, *node_r;
	int *edge_l, *edge_r, *tmp;
    int *NUM_L, *NUM_R, *NUM_EDGES, _;
    int *num_mb;
    long long *time_section;
    // MBE
    int *u2L, *v2P, *v2Q, *L, *R, *P, *Q;
    int *x, *L_lp, *R_lp, *P_lp, *Q_lp;
    int *L_buf, *num_N_u, *pre_min;
    // MBE_82
    int *g_u2L, *g_v2P, *g_v2Q, *g_L, *g_R, *g_P, *g_Q;
    int *g_x, *g_L_lp, *g_R_lp, *g_P_lp, *g_Q_lp;
    int *g_L_buf, *g_num_N_u, *g_pre_min;
    int *ori_P, *ori_P1, *ori_Q1, *ori_L1;
    int *P_ptr1, *fix_P_ptr1, *fix_Q_ptr1;
    hipMallocManaged(&NUM_EDGES   , sizeof(int));
    hipMallocManaged(&NUM_L       , sizeof(int));
    hipMallocManaged(&NUM_R       , sizeof(int));
    hipMallocManaged(&num_mb      , sizeof(int));
    hipMallocManaged(&time_section, sizeof(long long)*NUM_CLK);
    *num_mb = 0;
    my_memset(time_section, (long long)0, NUM_CLK);

    ifstream fin;
    fin.open(argv[1]);
    fin >> *NUM_R >> *NUM_L >> *NUM_EDGES;
    hipMallocManaged(&tmp   , sizeof(int )*(*NUM_L    ));
    hipMallocManaged(&node_l, sizeof(Node)*(*NUM_L    ));
    hipMallocManaged(&edge_l, sizeof(int )*(*NUM_EDGES));
    hipMallocManaged(&node_r, sizeof(Node)*(*NUM_R    ));
    hipMallocManaged(&edge_r, sizeof(int )*(*NUM_EDGES));
    for (int i = 0; i < *NUM_R    ; i++) fin >> node_r[i].start >> node_r[i].length;
    for (int i = 0; i < *NUM_EDGES; i++) fin >> edge_r[i] >> _;
    fin.close();

    int numBlocksPerSM;
    int numThreads = NUM_THDS;
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_MBE_cuMBE, numThreads, 0);
    int numBlocks_max = deviceProp.multiProcessorCount * numBlocksPerSM;
    int numBlocks = NUM_BLKS > numBlocks_max ? numBlocks_max : \
                    NUM_BLKS > 0 ? NUM_BLKS : 1;
    dim3 num_blocks_TRANSPOSE(numBlocks, 1, 1);
    dim3 num_blocks_MBE(numBlocks, 1, 1);
    dim3 block_size(numThreads, 1, 1);

    bool swap_RL = *NUM_R > *NUM_L;
    if (swap_RL) {
        swap(*NUM_L, *NUM_R);
        swap(node_l, node_r);
        swap(edge_l, edge_r);
    }

    // MBE
    hipMallocManaged(&u2L    , sizeof(int)*(*NUM_L)); my_memset_order(u2L, 0, *NUM_L);
    hipMallocManaged(&v2P    , sizeof(int)*(*NUM_R)); my_memset_order(v2P, 0, *NUM_R);
    hipMallocManaged(&v2Q    , sizeof(int)*(*NUM_R)); my_memset_order(v2Q, 0, *NUM_R);
    hipMallocManaged(&L      , sizeof(int)*(*NUM_L)); my_memset_order(L  , 0, *NUM_L);
    hipMallocManaged(&R      , sizeof(int)*(*NUM_R)); my_memset_order(R  , 0, *NUM_R);
    hipMallocManaged(&P      , sizeof(int)*(*NUM_R)); my_memset_order(P  , 0, *NUM_R);
    hipMallocManaged(&Q      , sizeof(int)*(*NUM_R)); my_memset_order(Q  , 0, *NUM_R);
    hipMallocManaged(&x      , sizeof(int)*(*NUM_R)); my_memset(x   ,     -1, *NUM_R);
    hipMallocManaged(&L_lp   , sizeof(int)*(*NUM_R)); my_memset(L_lp, *NUM_L, *NUM_R);
    hipMallocManaged(&R_lp   , sizeof(int)*(*NUM_R)); my_memset(R_lp,      0, *NUM_R);
    hipMallocManaged(&P_lp   , sizeof(int)*(*NUM_R)); my_memset(P_lp, *NUM_R, *NUM_R);
    hipMallocManaged(&Q_lp   , sizeof(int)*(*NUM_R)); my_memset(Q_lp,      0, *NUM_R);
    hipMallocManaged(&L_buf  , sizeof(int)*(*NUM_L)); my_memset(L_buf,     0, *NUM_L);
    hipMallocManaged(&num_N_u, sizeof(int)*(*NUM_R)); my_memset(num_N_u,   0, *NUM_R);
    hipMallocManaged(&pre_min, sizeof(int)*(*NUM_R)); my_memset(pre_min,   1, *NUM_R);
    // MBE_82
    hipMallocManaged(&g_u2L    , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )     g_u2L[i] =     u2L[i % (*NUM_L)];
    hipMallocManaged(&g_v2P    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_v2P[i] =     v2P[i % (*NUM_R)];
    hipMallocManaged(&g_v2Q    , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )     g_v2Q[i] =     v2Q[i % (*NUM_R)];
    hipMallocManaged(&g_L      , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )       g_L[i] =       L[i % (*NUM_L)];
    hipMallocManaged(&g_R      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_R[i] =       R[i % (*NUM_R)];
    hipMallocManaged(&g_P      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_P[i] =       P[i % (*NUM_R)];
    hipMallocManaged(&g_Q      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_Q[i] =       Q[i % (*NUM_R)];
    hipMallocManaged(&g_x      , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )       g_x[i] =       x[i % (*NUM_R)];
    hipMallocManaged(&g_L_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_L_lp[i] =    L_lp[i % (*NUM_R)];
    hipMallocManaged(&g_R_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_R_lp[i] =    R_lp[i % (*NUM_R)];
    hipMallocManaged(&g_P_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_P_lp[i] =    P_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_lp   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_Q_lp[i] =    Q_lp[i % (*NUM_R)];
    hipMallocManaged(&g_L_buf  , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )   g_L_buf[i] =   L_buf[i % (*NUM_L)];
    hipMallocManaged(&g_num_N_u, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_num_N_u[i] = num_N_u[i % (*NUM_R)];
    hipMallocManaged(&g_pre_min, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_pre_min[i] = pre_min[i % (*NUM_R)];
    hipMallocManaged(&ori_P     , sizeof(int)*(*NUM_R));
    hipMallocManaged(&ori_P1    , sizeof(int)*(*NUM_R)*numBlocks);
    hipMallocManaged(&ori_Q1    , sizeof(int)*(*NUM_R)*numBlocks);
    hipMallocManaged(&ori_L1    , sizeof(int)*(*NUM_L)*numBlocks);
    hipMallocManaged(&P_ptr1    , sizeof(int)         *numBlocks);
    hipMallocManaged(&fix_P_ptr1, sizeof(int)         *numBlocks);
    hipMallocManaged(&fix_Q_ptr1, sizeof(int)         *numBlocks);

    void *kernelArgs_CSR2CSC[] = {&tmp, &node_r, &edge_r, &node_l, &edge_l, &NUM_R, &NUM_L, &NUM_EDGES};
    void *kernelArgs_CSC2CSR[] = {&tmp, &node_l, &edge_l, &node_r, &edge_r, &NUM_L, &NUM_R, &NUM_EDGES};
    void *kernelArgs_MBE[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node_l, &edge_l, &node_r, &edge_r,
                              &g_u2L, &g_v2P, &g_v2Q, &g_L, &g_R, &g_P, &g_Q, &g_x, &g_L_lp, &g_R_lp, &g_P_lp, &g_Q_lp, &g_L_buf, &g_num_N_u, &g_pre_min,
                              &ori_P, &ori_P1, &ori_Q1, &ori_L1, &P_ptr1, &fix_P_ptr1, &fix_Q_ptr1, &num_mb, &time_section};

    string algo = ALGORITHM;
    algo = algo == "noRS" || algo == "noES" || algo == "noWS" ? algo : "cuMBE";
    string filename = dataset.substr(0, dataset.rfind('.')) + "_" + algo;
    
    ofstream fout;
    fout.open("result/"+filename);

#ifdef DEBUG
    cout << "\33[2J\33[1;1H";
#endif /* DEBUG */
    cout << "date/time: "<< ctime(&tmNow);
    cout << "algorithm: " << algo << endl;
    cout << "dataset: " << dataset << endl;
    cout << "|R|: " << *NUM_R << ", |L|: " << *NUM_L << ", |E|: " << *NUM_EDGES << endl;
    cout << "grid_size: " << numBlocks << ", block_size: " << numThreads << endl;

    fout << "date/time: "<< ctime(&tmNow);
    fout << "algorithm: " << algo << endl;
    fout << "dataset: " << dataset << endl;
    fout << "|R|: " << *NUM_R << ", |L|: " << *NUM_L << ", |E|: " << *NUM_EDGES << endl;
    fout << "grid_size: " << numBlocks << ", block_size: " << numThreads << endl;

    hipLaunchCooperativeKernel((void*)CUDA_TRANSPOSE, num_blocks_TRANSPOSE, block_size, swap_RL ? kernelArgs_CSC2CSR : kernelArgs_CSR2CSC);
    hipDeviceSynchronize();
    my_memset_sort(ori_P, 0, *NUM_R, node_r);
    
    hipMemPrefetchAsync(&NUM_EDGES   , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&NUM_L       , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&NUM_R       , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&num_mb      , sizeof(int), device, NULL);
    hipMemPrefetchAsync(&time_section, sizeof(long long)*NUM_CLK, device, NULL);

    hipMemPrefetchAsync(&node_l, sizeof(Node)*(*NUM_L    ), device, NULL);
    hipMemPrefetchAsync(&edge_l, sizeof(int )*(*NUM_EDGES), device, NULL);
    hipMemPrefetchAsync(&node_r, sizeof(Node)*(*NUM_R    ), device, NULL);
    hipMemPrefetchAsync(&edge_r, sizeof(int )*(*NUM_EDGES), device, NULL);
    
    hipMemPrefetchAsync(g_u2L     , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_v2P     , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_v2Q     , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_L       , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_R       , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_P       , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_Q       , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_x       , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_L_lp    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_R_lp    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_P_lp    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_Q_lp    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_L_buf   , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_num_N_u , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(g_pre_min , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(ori_P     , sizeof(int)*(*NUM_R)          , device, NULL);
    hipMemPrefetchAsync(ori_P1    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(ori_Q1    , sizeof(int)*(*NUM_R)*numBlocks, device, NULL);
    hipMemPrefetchAsync(ori_L1    , sizeof(int)*(*NUM_L)*numBlocks, device, NULL);
    hipMemPrefetchAsync(P_ptr1    , sizeof(int)         *numBlocks, device, NULL);
    hipMemPrefetchAsync(fix_P_ptr1, sizeof(int)         *numBlocks, device, NULL);
    hipMemPrefetchAsync(fix_Q_ptr1, sizeof(int)         *numBlocks, device, NULL);

    hipDeviceSynchronize();

    int stat;
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if      (algo == "noRS")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_noRS , num_blocks_MBE, block_size, kernelArgs_MBE);
    else if (algo == "noES")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_noES , num_blocks_MBE, block_size, kernelArgs_MBE);
    else if (algo == "noWS")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_noWS , num_blocks_MBE, block_size, kernelArgs_MBE);
    else if (algo == "cuMBE")
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_cuMBE, num_blocks_MBE, block_size, kernelArgs_MBE);
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    // cout << "status: " << stat << endl;
    cout << "maximal bicliques: " << *num_mb << endl;
#ifdef DESECTION
    long long sum_time_section = 0;
    for (int i = 0; i < NUM_CLK; i++)
        sum_time_section += time_section[i];
    cout << "time percentage:" << fixed << setprecision(4);
    for (int i = 0; i < NUM_CLK; i++)
        cout << " " << (double)time_section[i] * 100 / sum_time_section;
    cout << setprecision(6) << endl;
    // cout << " " << (double)time_section[8] / 1932735283.2 / 246 << "\n";
    // cout << "runtime (s) (no L): " << (time * ((double)(sum_time_section - time_section[10])) / sum_time_section)/1000 << endl;
#endif /* DESECTION */
#ifdef DEBUG
    cout << "runtime (s): " << time/1000 << endl;
    cout << "\33[" << (numBlocks-1) / WORDS_1ROW + 9 << ";1H";
#else  /* DEBUG */
    cout << "runtime (s): " << time/1000 << endl;
#endif /* DEBUG */

    fout << "maximal bicliques: " << *num_mb << endl;
#ifdef DESECTION
    fout << "time percentage:" << fixed << setprecision(4);
    for (int i = 0; i < NUM_CLK; i++)
        fout << " " << (double)time_section[i] * 100 / sum_time_section;
    fout << setprecision(6) << endl;
    // fout << "runtime (s) (no L): " << (time * ((double)(sum_time_section - time_section[10])) / sum_time_section)/1000 << endl;
#endif /* DESECTION */
    fout << "runtime (s): " << time/1000 << endl;

    hipFree(tmp);
    hipFree(node_l);
    hipFree(edge_l);
    hipFree(node_r);
    hipFree(edge_r);
    hipFree(NUM_L);
    hipFree(NUM_R);
    hipFree(NUM_EDGES);
    hipFree(num_mb);
    hipFree(time_section);
    // MBE
    hipFree(u2L);
    hipFree(v2P);
    hipFree(v2Q);
    hipFree(L);
    hipFree(R);
    hipFree(P);
    hipFree(Q);
    hipFree(x);
    hipFree(L_lp);
    hipFree(R_lp);
    hipFree(P_lp);
    hipFree(Q_lp);
    hipFree(L_buf);
    hipFree(num_N_u);
    hipFree(pre_min);
    // MBE_82
    hipFree(g_u2L);
    hipFree(g_v2P);
    hipFree(g_v2Q);
    hipFree(g_L);
    hipFree(g_R);
    hipFree(g_P);
    hipFree(g_Q);
    hipFree(g_x);
    hipFree(g_L_lp);
    hipFree(g_R_lp);
    hipFree(g_P_lp);
    hipFree(g_Q_lp);
    hipFree(g_L_buf);
    hipFree(g_num_N_u);
    hipFree(g_pre_min);
    hipFree(ori_P);
    hipFree(ori_P1);
    hipFree(ori_Q1);
    hipFree(ori_L1);
    hipFree(P_ptr1);
    hipFree(fix_P_ptr1);
    hipFree(fix_Q_ptr1);
}