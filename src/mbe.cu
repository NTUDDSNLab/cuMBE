#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <bits/stdc++.h>
#include <hip/hip_cooperative_groups.h>
#include <sys/ioctl.h>
using namespace std;
using namespace std::chrono;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define INF  1073741824
#define ONE  1
#define ZERO 0
#define NUM_THDS 256
#define NUM_BLKS atoi(argv[2])
#define LOG_BLK_ID 1
#define CLK(IDX) if (!threadIdx.x) { clk[IDX] += clock() - clk_; clk_ = clock(); }
#define CLK_CPU(IDX) clk[IDX] += clock() - clk_; clk_ = clock();
// #define CLK(IDX) ;
// #define CLK_CPU(IDX) ;

typedef struct {
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

typedef struct {
	unordered_set<int> L;
	unordered_set<int> R;
} Biclique;

template <class T>
void my_memset(T *SA, T val, int len) {
    for (int i = 0; i < len; i++)
        SA[i] = val;
}

void my_memset_order(int *SA, int val_start, int val_end) {
    for (int i = val_start; i < val_end; i++)
        SA[i - val_start] = i;
}

void maximal_bic_enum_set(int *NUM_L, int *NUM_R, int *NUM_EDGES, Node *node, int *edge,
                          int *u2L, int *L, int *R, int *P, int *Q,
                          int *x, int *L_lp, int *R_lp, int *P_lp, int *Q_lp) {
    vector<Biclique> maximal_bicliques;
    int num_maximal_bicliques = 0;
    vector< unordered_set<int> > Q_set(*NUM_R);
    Q_set[0].clear();

    long long clk[10] = { 0 }, clk_ = clock();
    
    for (int lvl = 0; lvl >= 0; ) {

        // printf("lvl: %d\n", lvl);

        int *x_cur    = &(   x[lvl]);
        int *L_lp_cur = &(L_lp[lvl]), *L_lp_nxt = &(L_lp[lvl+1]);
        int *R_lp_cur = &(R_lp[lvl]), *R_lp_nxt = &(R_lp[lvl+1]);
        int *P_lp_cur = &(P_lp[lvl]), *P_lp_nxt = &(P_lp[lvl+1]);
        unordered_set<int> *Q_cur = &(Q_set[lvl]), *Q_nxt = &(Q_set[lvl+1]);
        bool is_recursive = false;

        // while P ≠ ∅ do
        while (*P_lp_cur != 0) {

            CLK_CPU(0);

            //string tab_level(lvl << 3, ' ');
            //printf("\n%sL:", tab_level.c_str());
            //for (int i = 0; i < *L_lp_cur; i++)
            //    printf(" %d", L[i]);
            //printf("\n%sR:", tab_level.c_str());
            //for (int i = 0; i < *R_lp_cur; i++)
            //    printf(" %d", R[i]);
            //printf("\n%sP:", tab_level.c_str());
            //for (int i = 0; i < *P_lp_cur; i++)
            //    printf(" %d", P[i]);
            //printf("\n%sQ:", tab_level.c_str());
            //for (int i = 0; i < *Q_lp_cur; i++)
            //    printf(" %d", Q[i]);

            // Select x from P;
            // P <--- P \ {x};
            *x_cur = P[--(*P_lp_cur)];
            //// printf("x: %d\n", *x_cur);
            
            // R' <--- R ∪ {x};
            *R_lp_nxt = *R_lp_cur;
            R[(*R_lp_nxt)++] = *x_cur;

            CLK_CPU(1);

            *L_lp_nxt = 0; // |L'|

            CLK_CPU(2);

            // L' <--- {u ∈ L | (u, x) ∈ E(G)};
            for (int eid = node[*x_cur].start, eid_end = eid + node[*x_cur].length; eid < eid_end; eid++) {
                int u = edge[eid];
                int l = u2L[u];
                if (l < *L_lp_cur) {
                    swap(L[(*L_lp_nxt)++], L[l]);
                    swap(u2L[L[l]], u2L[u]);
                }
            }

            CLK_CPU(3);

            //// printf("L':");
            //// for (int i = 0; i < *L_lp_nxt; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n");
            
            // P' ← ∅; Q' ← ∅;
            *P_lp_nxt = 0; (*Q_nxt).clear();

            bool is_maximal = true;

            // foreach v ∈ Q
            for (const auto &v : *Q_cur) {

                int num_N_v = 0; // |N[v]|
                // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                for (int eid = node[v].start, eid_end = eid + node[v].length; eid < eid_end; eid++) {
                    int u = edge[eid];
                    int l = u2L[u];
                    if (l < *L_lp_nxt)
                        num_N_v++;
                }
                
                // if |N[v]| = |L'| then
                if (num_N_v == *L_lp_nxt) {
                    is_maximal = false;
                    break;
                }
                // else if |N[v]| > 0 then
                else if (num_N_v > 0)
                    // Q' ← Q' ∪ {v};
                    (*Q_nxt).insert(v);
                
            }

            CLK_CPU(4);

            //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

            // if is_maximal = TRUE then
            if (is_maximal == true) {

                // foreach v ∈ P do
                for (int i = 0; i < *P_lp_cur; i++) {
                    int v = P[i];

                    int num_N_v = 0; // |N[v]|
                    // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                    for (int eid = node[v].start, eid_end = eid + node[v].length; eid < eid_end; eid++) {
                        int u = edge[eid];
                        int l = u2L[u];
                        if (l < *L_lp_nxt)
                            num_N_v++;
                    }
                    
                    // if |N[v]| = |L'| then
                    if (num_N_v == *L_lp_nxt)
                        // R' ← R' ∪ {v};
                        R[(*R_lp_nxt)++] = v;
                    // else if |N[v]| > 0 then
                    else if (num_N_v > 0)
                        // P' ← P' ∪ {v};
                        swap(P[(*P_lp_nxt)++], P[i]);

                }

                CLK_CPU(5);
                
                //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

                // PRINT(L', R');
                //                printf("\n> Find maximal biclique (No. %d)", num_maximal_bicliques++);
                //                printf("\nL':");
                //                for (int i = 0; i < *L_lp_nxt; i++)
                //                    printf(" %d", L[i]);
                //                printf("\nR':");
                //                for (int i = 0; i < *R_lp_nxt; i++)
                //                    printf(" %d", R[i]);
                //                printf("\n");
                //
                // save maximal bicliques
                //// Biclique new_maximal_bicliques;
                //// for (int i = 0; i < *L_lp_nxt; i++)
                ////     new_maximal_bicliques.L.insert(L[i]);
                //// for (int i = 0; i < *R_lp_nxt; i++)
                ////     new_maximal_bicliques.R.insert(R[i]);
                //// maximal_bicliques.push_back(new_maximal_bicliques);
                if (++num_maximal_bicliques << 22 == 0)
                    printf("%d\n", num_maximal_bicliques);

                CLK_CPU(6);

                // if P' ≠ ∅ then
                if (*P_lp_nxt != 0) {
                    // biclique_find(G, L', R', P', Q');
                    //// printf("\n往 下 安安");
                    lvl++;
                    is_recursive = true;
                    break;
                }

            }
            else {
                //// printf("\n不安安");
            }

            // Q ← Q ∪ {x};
            (*Q_cur).insert(*x_cur);
            //// printf("\n往 右 安安");
        }

        if (!is_recursive) {
            if (lvl--)
                Q_set[lvl].insert(x[lvl]);
            //// printf("\n往 上 安安");
            //// printf("\n往 右 安安");
        }

    }

    printf("\nFind %d maximal bicliques.\n", num_maximal_bicliques);
    printf("time:");
    for (int i = 0; i < 10; i++) {
        // clk[i] >>= 21;
        printf(" %lld", clk[i]);
    }
    printf("\n");

    if (*NUM_R > 20 || *NUM_L > 20) return;

    string _ = "";
    printf("\33[2J\33[0;0H");

    printf("  ");
    for (int i = 0; i < *NUM_L; i++)
        printf(" %d", i / 10);
    printf("\n  ");
    for (int i = 0; i < *NUM_L; i++)
        printf(" %d", i % 10);
    printf("\n");
    for (int i = 0; i < *NUM_R; i++) {
        bool adj_vec[*NUM_L] = { false };
        for (int j = node[i].start, j_end = j + node[i].length; j < j_end; j++)
            adj_vec[edge[j]] = true;
        printf("%d%d", i / 10, i % 10);
        for (int j = 0; j < *NUM_L; j++)
            printf(" %c", adj_vec[j] ? '#' : '-');
        printf("\n");
    }

    for (int i = 0, i_end = maximal_bicliques.size(); i < i_end; i++) {
        printf("\33[7m");
        for (const auto &r: maximal_bicliques[i].R)
            for (const auto &l: maximal_bicliques[i].L)
                printf("\33[%d;%dH#\n", 3 + r, 4 + l * 2);
        printf("\33[0m\n\33[%d;0H\n", 3 + (*NUM_R));
        if      (_ == "auto") usleep(800000);
        else if (_ != "exit") cin >> _;
        for (const auto &r: maximal_bicliques[i].R)
            for (const auto &l: maximal_bicliques[i].L)
                printf("\33[%d;%dH#\n", 3 + r, 4 + l * 2);
    }
    printf("\33[%d;0H\n", 3 + (*NUM_R));
}

void maximal_bic_enum(int *NUM_L, int *NUM_R, int *NUM_EDGES, Node *node, int *edge,
                      int *u2L, int *L, int *R, int *P, int *Q,
                      int *x, int *L_lp, int *R_lp, int *P_lp, int *Q_lp) {
    vector<Biclique> maximal_bicliques;
    int num_maximal_bicliques = 0;

    long long clk[10] = { 0 }, clk_ = clock();
    
    for (int lvl = 0; lvl >= 0; ) {

        // printf("lvl: %d\n", lvl);

        int *x_cur    = &(   x[lvl]);
        int *L_lp_cur = &(L_lp[lvl]), *L_lp_nxt = &(L_lp[lvl+1]);
        int *R_lp_cur = &(R_lp[lvl]), *R_lp_nxt = &(R_lp[lvl+1]);
        int *P_lp_cur = &(P_lp[lvl]), *P_lp_nxt = &(P_lp[lvl+1]);
        int *Q_lp_cur = &(Q_lp[lvl]), *Q_lp_nxt = &(Q_lp[lvl+1]);
        bool is_recursive = false;

        // while P ≠ ∅ do
        while (*P_lp_cur != 0) {

            CLK_CPU(0);

            //string tab_level(lvl << 3, ' ');
            //printf("\n%sL:", tab_level.c_str());
            //for (int i = 0; i < *L_lp_cur; i++)
            //    printf(" %d", L[i]);
            //printf("\n%sR:", tab_level.c_str());
            //for (int i = 0; i < *R_lp_cur; i++)
            //    printf(" %d", R[i]);
            //printf("\n%sP:", tab_level.c_str());
            //for (int i = 0; i < *P_lp_cur; i++)
            //    printf(" %d", P[i]);
            //printf("\n%sQ:", tab_level.c_str());
            //for (int i = 0; i < *Q_lp_cur; i++)
            //    printf(" %d", Q[i]);

            // Select x from P;
            // P <--- P \ {x};
            *x_cur = P[--(*P_lp_cur)];
            //// printf("x: %d\n", *x_cur);
            
            // R' <--- R ∪ {x};
            *R_lp_nxt = *R_lp_cur;
            R[(*R_lp_nxt)++] = *x_cur;

            CLK_CPU(1);

            *L_lp_nxt = 0; // |L'|

            CLK_CPU(2);

            // L' <--- {u ∈ L | (u, x) ∈ E(G)};
            for (int eid = node[*x_cur].start, eid_end = eid + node[*x_cur].length; eid < eid_end; eid++) {
                int u = edge[eid];
                int l = u2L[u];
                if (l < *L_lp_cur) {
                    swap(L[(*L_lp_nxt)++], L[l]);
                    swap(u2L[L[l]], u2L[u]);
                }
            }

            CLK_CPU(3);

            //// printf("L':");
            //// for (int i = 0; i < *L_lp_nxt; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n");
            
            // P' ← ∅; Q' ← ∅;
            *P_lp_nxt = 0; *Q_lp_nxt = *Q_lp_cur;

            bool is_maximal = true;

            // foreach v ∈ Q
            for (int i = 0; i < *Q_lp_cur; i++) {

                int v = Q[i];

                int num_N_v = 0; // |N[v]|
                // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                for (int eid = node[v].start, eid_end = eid + node[v].length; eid < eid_end; eid++) {
                    int u = edge[eid];
                    int l = u2L[u];
                    if (l < *L_lp_nxt)
                        num_N_v++;
                }
                
                // if |N[v]| = |L'| then
                if (num_N_v == *L_lp_nxt) {
                    is_maximal = false;
                    break;
                }
                // // else if |N[v]| > 0 then
                // else if (num_N_v == 0)
                //     // Q' ← Q' ∪ {v};
                //     swap(Q[(*Q_ls_nxt)++], Q[i]);
                
            }

            CLK_CPU(4);

            //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

            // if is_maximal = TRUE then
            if (is_maximal == true) {

                // foreach v ∈ P do
                for (int i = 0; i < *P_lp_cur; i++) {
                    int v = P[i];

                    int num_N_v = 0; // |N[v]|
                    // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                    for (int eid = node[v].start, eid_end = eid + node[v].length; eid < eid_end; eid++) {
                        int u = edge[eid];
                        int l = u2L[u];
                        if (l < *L_lp_nxt)
                            num_N_v++;
                    }
                    
                    // if |N[v]| = |L'| then
                    if (num_N_v == *L_lp_nxt)
                        // R' ← R' ∪ {v};
                        R[(*R_lp_nxt)++] = v;
                    // else if |N[v]| > 0 then
                    else if (num_N_v > 0)
                        // P' ← P' ∪ {v};
                        swap(P[(*P_lp_nxt)++], P[i]);

                }

                CLK_CPU(5);
                
                //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

                // PRINT(L', R');
                //                printf("\n> Find maximal biclique (No. %d)", num_maximal_bicliques++);
                //                printf("\nL':");
                //                for (int i = 0; i < *L_lp_nxt; i++)
                //                    printf(" %d", L[i]);
                //                printf("\nR':");
                //                for (int i = 0; i < *R_lp_nxt; i++)
                //                    printf(" %d", R[i]);
                //                printf("\n");
                //
                // save maximal bicliques
                //// Biclique new_maximal_bicliques;
                //// for (int i = 0; i < *L_lp_nxt; i++)
                ////     new_maximal_bicliques.L.insert(L[i]);
                //// for (int i = 0; i < *R_lp_nxt; i++)
                ////     new_maximal_bicliques.R.insert(R[i]);
                //// maximal_bicliques.push_back(new_maximal_bicliques);
                if (++num_maximal_bicliques << 22 == 0)
                    printf("%d\n", num_maximal_bicliques);

                CLK_CPU(6);

                // if P' ≠ ∅ then
                if (*P_lp_nxt != 0) {
                    // biclique_find(G, L', R', P', Q');
                    //// printf("\n往 下 安安");
                    lvl++;
                    is_recursive = true;
                    break;
                }

            }
            else {
                //// printf("\n不安安");
            }

            // Q ← Q ∪ {x};
            Q[(*Q_lp_cur)++] = *x_cur;
            //// printf("\n往 右 安安");
        }

        if (!is_recursive) {
            lvl--;
            Q[Q_lp[lvl]++] = x[lvl];
            //// printf("\n往 上 安安");
            //// printf("\n往 右 安安");
        }

    }

    printf("\nFind %d maximal bicliques.\n", num_maximal_bicliques);
    printf("time:");
    for (int i = 0; i < 10; i++) {
        // clk[i] >>= 21;
        printf(" %lld", clk[i]);
    }
    printf("\n");

    if (*NUM_R > 20 || *NUM_L > 20) return;

    string _ = "";
    printf("\33[2J\33[0;0H");

    printf("  ");
    for (int i = 0; i < *NUM_L; i++)
        printf(" %d", i / 10);
    printf("\n  ");
    for (int i = 0; i < *NUM_L; i++)
        printf(" %d", i % 10);
    printf("\n");
    for (int i = 0; i < *NUM_R; i++) {
        bool adj_vec[*NUM_L] = { false };
        for (int j = node[i].start, j_end = j + node[i].length; j < j_end; j++)
            adj_vec[edge[j]] = true;
        printf("%d%d", i / 10, i % 10);
        for (int j = 0; j < *NUM_L; j++)
            printf(" %c", adj_vec[j] ? '#' : '-');
        printf("\n");
    }

    for (int i = 0, i_end = maximal_bicliques.size(); i < i_end; i++) {
        printf("\33[7m");
        for (const auto &r: maximal_bicliques[i].R)
            for (const auto &l: maximal_bicliques[i].L)
                printf("\33[%d;%dH#\n", 3 + r, 4 + l * 2);
        printf("\33[0m\n\33[%d;0H\n", 3 + (*NUM_R));
        if      (_ == "auto") usleep(800000);
        else if (_ != "exit") cin >> _;
        for (const auto &r: maximal_bicliques[i].R)
            for (const auto &l: maximal_bicliques[i].L)
                printf("\33[%d;%dH#\n", 3 + r, 4 + l * 2);
    }
    printf("\33[%d;0H\n", 3 + (*NUM_R));
}

__global__ void CUDA_MBE(int *NUM_L, int *NUM_R, int *NUM_EDGES, Node *node, int *edge,
                         int *u2L, int *L, int *R, int *P, int *Q,
                         int *x, int *L_lp, int *R_lp, int *P_lp, int *Q_lp) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_total_thds = gridDim.x * blockDim.x;
    int wid = threadIdx.x >> 5;
    int lid = threadIdx.x & 0x1f;
    int num_warps = blockDim.x >> 5;
    int num_maximal_bicliques = 0;
    __shared__ int lvl;
    __shared__ int *x_cur;
    __shared__ int *L_lp_cur, *L_lp_nxt;
    __shared__ int *R_lp_cur, *R_lp_nxt;
    __shared__ int *P_lp_cur, *P_lp_nxt;
    __shared__ int *Q_lp_cur, *Q_lp_nxt;
    __shared__ bool is_recursive;
    __shared__ bool is_maximal;
    __shared__ int num_L_nxt, num_N_v;
    // __shared__ int num_N_v[32];

    __shared__ long long clk[10], clk_;

    if (!threadIdx.x) {
        clk_ = clock();
        for (int i = 0; i < 10; i++)
            clk[i] = 0;
    }

    if (!threadIdx.x)
        lvl = 0;

    __syncthreads();

    for (; lvl >= 0; ) {

        // if (!threadIdx.x)
        //     printf("\nlvl: %d", lvl);

        x_cur    = &(   x[lvl]);
        L_lp_cur = &(L_lp[lvl]); L_lp_nxt = &(L_lp[lvl+1]);
        R_lp_cur = &(R_lp[lvl]); R_lp_nxt = &(R_lp[lvl+1]);
        P_lp_cur = &(P_lp[lvl]); P_lp_nxt = &(P_lp[lvl+1]);
        Q_lp_cur = &(Q_lp[lvl]); Q_lp_nxt = &(Q_lp[lvl+1]);

        if (!threadIdx.x)
            is_recursive = false;
        
        __syncthreads();

        // while P ≠ ∅ do
        while (*P_lp_cur != 0) {

            CLK(0);

            if (!threadIdx.x) {

                // printf("\n");
                // for (int i = 0; i < lvl; i++) printf("        ");
                // printf("L:");
                // for (int i = 0; i < *NUM_L; i++)
                //     printf(" %d", L[i]);
                // printf("\n");
                // for (int i = 0; i < lvl; i++) printf("        ");
                // printf("R:");
                // for (int i = 0; i < *R_lp_cur; i++)
                //     printf(" %d", R[i]);
                // printf("\n");
                // for (int i = 0; i < lvl; i++) printf("        ");
                // printf("P:");
                // for (int i = 0; i < *P_lp_cur; i++)
                //     printf(" %d", P[i]);
                // printf("\n");
                // for (int i = 0; i < lvl; i++) printf("        ");
                // printf("Q:");
                // for (int i = 0; i < *Q_lp_cur; i++)
                //     printf(" %d", Q[i]);

                // Select x from P;
                // P <--- P \ {x};
                *x_cur = P[--(*P_lp_cur)];
                //// printf("x: %d\n", *x_cur);
                
                // R' <--- R ∪ {x};
                *R_lp_nxt = *R_lp_cur;
                R[(*R_lp_nxt)++] = *x_cur;

                //// *L_lp_nxt = 0;
                num_L_nxt = 0;
            }
            
            __syncthreads();

            CLK(1);

            // |L'|
            for (int l = tid; l < *NUM_L; l += num_total_thds)
                L[l] = L[l] > lvl ? lvl : L[l];

            __syncthreads();

            CLK(2);
            
            // L' <--- {u ∈ L | (u, x) ∈ E(G)};
            for (int eid = node[*x_cur].start + threadIdx.x, eid_end = node[*x_cur].start + node[*x_cur].length; eid < eid_end; eid += blockDim.x) {
                int l = edge[eid];
                if (L[l] == lvl) {
                    L[l]++;
                    atomicAdd(&num_L_nxt, 1);
                }
            }

            __syncthreads();

            CLK(3);

            //// printf("L':");
            //// for (int i = 0; i < *L_lp_nxt; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n");
            
            if (!threadIdx.x) {
                
                // P' ← ∅; Q' ← ∅;
                *P_lp_nxt = 0; *Q_lp_nxt = *Q_lp_cur;
                is_maximal = true;

            }

            // foreach v ∈ Q
            for (int i = 0; i < *Q_lp_cur; i++) {

                int v = Q[i];

                if (!threadIdx.x)
                    num_N_v = 0; // |N[v]|

                __syncthreads();

                // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                for (int eid = node[v].start + threadIdx.x, eid_end = node[v].start + node[v].length; eid < eid_end; eid += blockDim.x) {
                    int l = edge[eid];
                    if (L[l] > lvl)
                        atomicAdd(&num_N_v, 1);
                }

                __syncthreads();
                
                // if |N[v]| = |L'| then
                if (num_N_v == num_L_nxt) {
                    is_maximal = false;
                    break;
                }
                // // else if |N[v]| > 0 then
                // else if (num_N_v == 0)
                //     // Q' ← Q' ∪ {v};
                //     swap(Q[(*Q_ls_nxt)++], Q[i]);

                __syncthreads();
                
            }
            
            CLK(4);

            __syncthreads();

            //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

            // if is_maximal = TRUE then
            if (is_maximal == true) {

                // foreach v ∈ P do
                for (int i = 0; i < *P_lp_cur; i++) {
                    int v = P[i];

                    if (!threadIdx.x)
                        num_N_v = 0; // |N[v]|

                    __syncthreads();

                    // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                    for (int eid = node[v].start + threadIdx.x, eid_end = node[v].start + node[v].length; eid < eid_end; eid += blockDim.x) {
                        int l = edge[eid];
                        if (L[l] > lvl)
                            atomicAdd(&num_N_v, 1);
                    }

                    __syncthreads();
                    
                    if (!threadIdx.x) {

                        // if |N[v]| = |L'| then
                        if (num_N_v == num_L_nxt)
                            // R' ← R' ∪ {v};
                            R[(*R_lp_nxt)++] = v;
                        // else if |N[v]| > 0 then
                        else if (num_N_v > 0) {
                            // P' ← P' ∪ {v};
                            int P_tmp = P[*P_lp_nxt];
                            P[(*P_lp_nxt)++] = P[i];
                            P[i] = P_tmp;
                        }

                    }

                    __syncthreads();

                }
            
                CLK(5);

                if (!threadIdx.x) {
                
                    //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

                    // PRINT(L', R');
                    // printf("\n> Find maximal biclique (No. %d)", num_maximal_bicliques++);
                    // printf("\nL':");
                    // for (int i = 0; i < *NUM_L; i++)
                    //     if (L[i] > lvl)
                    //         printf(" %d", i);
                    // printf("\nR':");
                    // for (int i = 0; i < *R_lp_nxt; i++)
                    //     printf(" %d", R[i]);
                    // printf("\n");

                    // save maximal bicliques
                    //// Biclique new_maximal_bicliques;
                    //// for (int i = 0; i < *L_lp_nxt; i++)
                    ////     new_maximal_bicliques.L.insert(L[i]);
                    //// for (int i = 0; i < *R_lp_nxt; i++)
                    ////     new_maximal_bicliques.R.insert(R[i]);
                    //// maximal_bicliques.push_back(new_maximal_bicliques);
                    if (++num_maximal_bicliques << 22 == 0)
                        printf("%d\n", num_maximal_bicliques);

                    // if P' ≠ ∅ then
                    if (*P_lp_nxt != 0) {
                        // biclique_find(G, L', R', P', Q');
                        //// printf("\n往 下 安安");
                        lvl++;
                        is_recursive = true;
                    }

                }

                __syncthreads();
            
                CLK(6);

                if (is_recursive)
                    break;

            }
            else {
                //// printf("\n不安安");
            }

            if (!threadIdx.x) {

                // Q ← Q ∪ {x};
                Q[(*Q_lp_cur)++] = *x_cur;
                //// printf("\n往 右 安安");

            }

        }
        
        __syncthreads();

        // printf("tid: %d, lvl: %d\n", tid, lvl);

        if (!threadIdx.x) {

            if (!is_recursive) {
                lvl--;
                Q[Q_lp[lvl]++] = x[lvl];
                //// printf("\n往 上 安安");
                //// printf("\n往 右 安安");
            }

        }

        __syncthreads();

    }

    if (!threadIdx.x) {
        printf("\nFind %d maximal bicliques.\n", num_maximal_bicliques);
        printf("time:");
        for (int i = 0; i < 10; i++) {
            // clk[i] >>= 21;
            printf(" %lld", clk[i]);
        }
        printf("\n");
    }
}

__device__ int g_clk[10];
__device__ int total_bic;
__device__ int P_ptr;
__global__ void CUDA_MBE_82(int *NUM_L, int *NUM_R, int *NUM_EDGES, Node *node, int *edge,
                            int *g_u2L, int *g_L, int *g_R, int *g_P, int *g_Q, int *g_Q_rm,
                            int *g_x, int *g_L_lp, int *g_R_lp, int *g_P_lp, int *g_Q_lp) {

    int *u2L  = g_u2L  + blockIdx.x * (*NUM_L);
    int *L    = g_L    + blockIdx.x * (*NUM_L);
    int *R    = g_R    + blockIdx.x * (*NUM_R);
    int *P    = g_P    + blockIdx.x * (*NUM_R);
    int *Q    = g_Q    + blockIdx.x * (*NUM_R);
    int *x    = g_x    + blockIdx.x * (*NUM_R);
    int *L_lp = g_L_lp + blockIdx.x * (*NUM_R);
    int *R_lp = g_R_lp + blockIdx.x * (*NUM_R);
    int *P_lp = g_P_lp + blockIdx.x * (*NUM_R);
    int *Q_lp = g_Q_lp + blockIdx.x * (*NUM_R);
    int *Q_rm = g_Q_rm + blockIdx.x * (*NUM_R);
    grid_group grid = this_grid();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_total_thds = gridDim.x * blockDim.x;
    // int wid = threadIdx.x >> 5;
    // int lid = threadIdx.x & 0x1f;
    // int num_warps = blockDim.x >> 5;
    int num_maximal_bicliques = 0;
    __shared__ int lvl;
    __shared__ int *x_cur;
    __shared__ int *L_lp_cur, *L_lp_nxt;
    __shared__ int *R_lp_cur, *R_lp_nxt;
    __shared__ int *P_lp_cur, *P_lp_nxt;
    __shared__ int *Q_lp_cur, *Q_lp_nxt;
    __shared__ bool is_recursive;
    __shared__ bool is_maximal;
    __shared__ int num_L_nxt, num_N_v;

    __shared__ long long clk[10], clk_;
    if (!threadIdx.x) {
        clk_ = clock();
        for (int i = 0; i < 10; i++)
            clk[i] = 0;
        if (!blockIdx.x)
            for (int i = 0; i < 10; i++)
                g_clk[i] = 0;
    }

    if (!tid) {
        P_ptr = *NUM_R - 1;
        total_bic = 0;
    }

    grid.sync();

    if (!threadIdx.x) {
        lvl = 0;
        // P_lp[0] = *NUM_R + blockIdx.x;
        P_lp[0] = *NUM_R;
        //// printf("blk %d, u2L : %p\n", blockIdx.x, u2L );
        //// printf("blk %d, L   : %p\n", blockIdx.x, L   );
        //// printf("blk %d, R   : %p\n", blockIdx.x, R   );
        //// printf("blk %d, P   : %p\n", blockIdx.x, P   );
        //// printf("blk %d, Q   : %p\n", blockIdx.x, Q   );
        //// printf("blk %d, x   : %p\n", blockIdx.x, x   );
        //// printf("blk %d, L_lp: %p\n", blockIdx.x, L_lp);
        //// printf("blk %d, R_lp: %p\n", blockIdx.x, R_lp);
        //// printf("blk %d, P_lp: %p\n", blockIdx.x, P_lp);
        //// printf("blk %d, Q_lp: %p\n", blockIdx.x, Q_lp);
    }

    __syncthreads();

    for (; lvl >= 0; ) {

        // if (!threadIdx.x)
        //     printf("\nlvl: %d", lvl);

        if (!threadIdx.x) {
            x_cur    = &(   x[lvl]);
            L_lp_cur = &(L_lp[lvl]); L_lp_nxt = &(L_lp[lvl+1]);
            R_lp_cur = &(R_lp[lvl]); R_lp_nxt = &(R_lp[lvl+1]);
            P_lp_cur = &(P_lp[lvl]); P_lp_nxt = &(P_lp[lvl+1]);
            Q_lp_cur = &(Q_lp[lvl]); Q_lp_nxt = &(Q_lp[lvl+1]);
            is_recursive = false;
        }
        
        __syncthreads();

        if (lvl == 0)
        // while P ≠ ∅ do
        while (*P_lp_cur >= gridDim.x || 1) {
            
            __syncthreads();

            //// if (!threadIdx.x && blockIdx.x == LOG_BLK_ID) {
            ////     printf("\nblock_%d subtree_%d has found %d maximal bicliques now", blockIdx.x, *P_lp_cur - gridDim.x, num_maximal_bicliques);
            ////     // num_maximal_bicliques = 0;
            //// }
            CLK(0);

            if (!threadIdx.x) {

                //// if (blockIdx.x == LOG_BLK_ID) {
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("L:");
                ////     for (int i = 0; i < *NUM_L; i++)
                ////         printf(" %d", L[i]);
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("R:");
                ////     for (int i = 0; i < *R_lp_cur; i++)
                ////         printf(" %d", R[i]);
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("P:");
                ////     for (int i = 0; i < *P_lp_cur; i++)
                ////         printf(" %d", P[i]);
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("Q:");
                ////     for (int i = 0; i < *Q_lp_cur; i++)
                ////         printf(" %d", Q[i]);
                //// }

                // atomically get a new 1-level sub-tree
                // Q <--- Q ∪ {x before P_ptr};
                // for (int i = *P_lp_cur, i_end = *P_lp_cur -= gridDim.x; --i > i_end; ) {
                //     if (i < *NUM_R) {
                //         Q_rm[*Q_lp_cur] = INF;
                //         Q[(*Q_lp_cur)++] = i;
                //     }
                // }
                for (int i = *P_lp_cur, i_end = *P_lp_cur = atomicAdd(&P_ptr, -1); --i > i_end; ) {
                    if (i >= 0) {
                        Q_rm[*Q_lp_cur] = INF;
                        Q[(*Q_lp_cur)++] = i;
                    }
                }

                // printf("blk %d, P_lp_cur: %d\n", blockIdx.x, *P_lp_cur);

                // reset P to ordered
                for (int i = 0; i < *NUM_R; i++)
                    P[i] = i;

                // Select x from P;
                // P <--- P \ {x before P_ptr and x_cur};
                *x_cur = *P_lp_cur;
                //// printf("x: %d\n", *x_cur);
                
                // R' <--- R ∪ {x};
                *R_lp_nxt = *R_lp_cur;
                R[(*R_lp_nxt)++] = *x_cur;

                //// *L_lp_nxt = 0;
                num_L_nxt = 0;
            }
            
            __syncthreads();

            CLK(1);

            if (*P_lp_cur < 0) break;

            // |L'|
            // for (int l = tid; l < *NUM_L; l += num_total_thds)
            for (int l = threadIdx.x; l < *NUM_L; l += blockDim.x)
                L[l] = L[l] > lvl ? lvl : L[l];

            __syncthreads();

            CLK(2);
            
            // L' <--- {u ∈ L | (u, x) ∈ E(G)};
            for (int eid = node[*x_cur].start + threadIdx.x, eid_end = node[*x_cur].start + node[*x_cur].length; eid < eid_end; eid += blockDim.x) {
                int l = edge[eid];
                if (L[l] == lvl) {
                    L[l]++;
                    atomicAdd(&num_L_nxt, 1);
                }
            }

            __syncthreads();

            CLK(3);

            //// printf("L':");
            //// for (int i = 0; i < *L_lp_nxt; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n");
            
            if (!threadIdx.x) {
                
                // P' ← ∅; Q' ← ∅;
                *P_lp_nxt = 0; *Q_lp_nxt = *Q_lp_cur;
                is_maximal = true;

            }

            // foreach v ∈ Q
            for (int i = 0; i < *Q_lp_cur; i++) {

                if (Q_rm[i] < lvl) {
                    __syncthreads();
                    continue;
                }

                int v = Q[i];

                if (!threadIdx.x)
                    num_N_v = 0; // |N[v]|

                __syncthreads();

                // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                for (int eid = node[v].start + threadIdx.x, eid_end = node[v].start + node[v].length; eid < eid_end; eid += blockDim.x) {
                    int l = edge[eid];
                    if (L[l] > lvl)
                        atomicAdd(&num_N_v, 1);
                }

                __syncthreads();

                if (!threadIdx.x)
                    Q_rm[i] = INF;
                
                // if |N[v]| = |L'| then
                if (num_N_v == num_L_nxt) {
                    is_maximal = false;
                    break;
                }
                // else if |N[v]| > 0 then
                else if (num_N_v == 0)
                    // Q' ← Q' ∪ {v};
                    // swap(Q[(*Q_ls_nxt)++], Q[i]);
                    if (!threadIdx.x)
                        Q_rm[i] = lvl;

                __syncthreads();
                
            }
            
            CLK(4);

            __syncthreads();

            //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

            // if is_maximal = TRUE then
            if (is_maximal == true) {

                // foreach v ∈ P do
                for (int i = 0; i < *P_lp_cur; i++) {
                    int v = P[i];

                    if (!threadIdx.x)
                        num_N_v = 0; // |N[v]|

                    __syncthreads();

                    // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                    for (int eid = node[v].start + threadIdx.x, eid_end = node[v].start + node[v].length; eid < eid_end; eid += blockDim.x) {
                        int l = edge[eid];
                        if (L[l] > lvl)
                            atomicAdd(&num_N_v, 1);
                    }

                    __syncthreads();
                    
                    if (!threadIdx.x) {

                        // if |N[v]| = |L'| then
                        if (num_N_v == num_L_nxt)
                            // R' ← R' ∪ {v};
                            R[(*R_lp_nxt)++] = v;
                        // else if |N[v]| > 0 then
                        else if (num_N_v > 0) {
                            // P' ← P' ∪ {v};
                            int P_tmp = P[*P_lp_nxt];
                            P[(*P_lp_nxt)++] = P[i];
                            P[i] = P_tmp;
                        }

                    }

                    __syncthreads();

                }
            
                CLK(5);

                if (!threadIdx.x) {
                
                    //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

                    //// if (blockIdx.x == LOG_BLK_ID) {
                    ////     // PRINT(L', R');
                    ////     printf("\n> Find maximal biclique (No. %d)", num_maximal_bicliques);
                    ////     printf("\nL':");
                    ////     for (int i = 0; i < *NUM_L; i++)
                    ////         if (L[i] > lvl)
                    ////             printf(" %d", i);
                    ////     printf("\nR':");
                    ////     for (int i = 0; i < *R_lp_nxt; i++)
                    ////         printf(" %d", R[i]);
                    ////     printf("\n");
                    //// }

                    // save maximal bicliques
                    //// Biclique new_maximal_bicliques;
                    //// for (int i = 0; i < *L_lp_nxt; i++)
                    ////     new_maximal_bicliques.L.insert(L[i]);
                    //// for (int i = 0; i < *R_lp_nxt; i++)
                    ////     new_maximal_bicliques.R.insert(R[i]);
                    //// maximal_bicliques.push_back(new_maximal_bicliques);

                    // if (++num_maximal_bicliques > 0)
                    //     printf("blk %d : %d\n", blockIdx.x, num_maximal_bicliques);
                    printf("\33[%d;%dH%d\n", blockIdx.x / 10 + 1, (blockIdx.x % 10) * 10 + 1, ++num_maximal_bicliques);

                    // if P' ≠ ∅ then
                    if (*P_lp_nxt != 0) {
                        // biclique_find(G, L', R', P', Q');
                        //// printf("\n往 下 安安");
                        lvl++;
                        is_recursive = true;
                    }

                }

                __syncthreads();
            
                CLK(6);

                if (is_recursive)
                    break;

            }
            else {
                //// printf("\n不安安");
            }

            if (!threadIdx.x) {

                // Q ← Q ∪ {x};
                Q_rm[*Q_lp_cur] = INF;
                Q[(*Q_lp_cur)++] = *x_cur;
                //// printf("\n往 右 安安");

            }

        }

        else // lvl >= 1
        // while P ≠ ∅ do
        while (*P_lp_cur != 0) {
            
            __syncthreads();

            CLK(0);

            if (!threadIdx.x) {

                //// if (blockIdx.x == LOG_BLK_ID) {
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("L:");
                ////     for (int i = 0; i < *NUM_L; i++)
                ////         printf(" %d", L[i]);
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("R:");
                ////     for (int i = 0; i < *R_lp_cur; i++)
                ////         printf(" %d", R[i]);
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("P:");
                ////     for (int i = 0; i < *P_lp_cur; i++)
                ////         printf(" %d", P[i]);
                ////     printf("\n");
                ////     for (int i = 0; i < lvl; i++) printf("        ");
                ////     printf("Q:");
                ////     for (int i = 0; i < *Q_lp_cur; i++)
                ////         printf(" %d", Q[i]);
                //// }

                // Select x from P;
                // P <--- P \ {x};
                *x_cur = P[--(*P_lp_cur)];
                //// printf("x: %d\n", *x_cur);
                
                // R' <--- R ∪ {x};
                *R_lp_nxt = *R_lp_cur;
                R[(*R_lp_nxt)++] = *x_cur;

                //// *L_lp_nxt = 0;
                num_L_nxt = 0;
            }
            
            __syncthreads();

            CLK(1);

            // |L'|
            for (int l = threadIdx.x; l < *NUM_L; l += blockDim.x)
                L[l] = L[l] > lvl ? lvl : L[l];

            __syncthreads();

            CLK(2);
            
            // L' <--- {u ∈ L | (u, x) ∈ E(G)};
            for (int eid = node[*x_cur].start + threadIdx.x, eid_end = node[*x_cur].start + node[*x_cur].length; eid < eid_end; eid += blockDim.x) {
                int l = edge[eid];
                if (L[l] == lvl) {
                    L[l]++;
                    atomicAdd(&num_L_nxt, 1);
                }
            }

            __syncthreads();

            CLK(3);

            //// printf("L':");
            //// for (int i = 0; i < *L_lp_nxt; i++)
            ////     printf(" %d", L[i]);
            //// printf("\n");
            
            if (!threadIdx.x) {
                
                // P' ← ∅; Q' ← ∅;
                *P_lp_nxt = 0; *Q_lp_nxt = *Q_lp_cur;
                is_maximal = true;

            }

            // foreach v ∈ Q
            for (int i = 0; i < *Q_lp_cur; i++) {
                if (Q_rm[i] < lvl) {
                    __syncthreads();
                    continue;
                }

                int v = Q[i];

                if (!threadIdx.x)
                    num_N_v = 0; // |N[v]|

                __syncthreads();

                // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                for (int eid = node[v].start + threadIdx.x, eid_end = node[v].start + node[v].length; eid < eid_end; eid += blockDim.x) {
                    int l = edge[eid];
                    if (L[l] > lvl)
                        atomicAdd(&num_N_v, 1);
                }
                
                __syncthreads();
                
                if (!threadIdx.x)
                    Q_rm[i] = INF;

                // if |N[v]| = |L'| then
                if (num_N_v == num_L_nxt) {
                    is_maximal = false;
                    break;
                }
                // else if |N[v]| > 0 then
                else if (num_N_v == 0)
                    // Q' ← Q' ∪ {v};
                    // swap(Q[(*Q_ls_nxt)++], Q[i]);
                    if (!threadIdx.x)
                        Q_rm[i] = lvl;

                __syncthreads();
                
            }
            
            CLK(4);

            __syncthreads();

            //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

            // if is_maximal = TRUE then
            if (is_maximal == true) {

                // foreach v ∈ P do
                for (int i = 0; i < *P_lp_cur; i++) {
                    int v = P[i];

                    if (!threadIdx.x)
                        num_N_v = 0; // |N[v]|

                    __syncthreads();

                    // N[v] ← {u ∈ L' | (u, v) ∈ E(G)};
                    for (int eid = node[v].start + threadIdx.x, eid_end = node[v].start + node[v].length; eid < eid_end; eid += blockDim.x) {
                        int l = edge[eid];
                        if (L[l] > lvl)
                            atomicAdd(&num_N_v, 1);
                    }

                    __syncthreads();
                    
                    if (!threadIdx.x) {

                        // if |N[v]| = |L'| then
                        if (num_N_v == num_L_nxt)
                            // R' ← R' ∪ {v};
                            R[(*R_lp_nxt)++] = v;
                        // else if |N[v]| > 0 then
                        else if (num_N_v > 0) {
                            // P' ← P' ∪ {v};
                            int P_tmp = P[*P_lp_nxt];
                            P[(*P_lp_nxt)++] = P[i];
                            P[i] = P_tmp;
                        }

                    }

                    __syncthreads();

                }
            
                CLK(5);

                if (!threadIdx.x) {
                
                    //// printf("R_lp_nxt: %d\n", *R_lp_nxt);

                    //// if (blockIdx.x == LOG_BLK_ID) {
                    ////     // PRINT(L', R');
                    ////     printf("\n> Find maximal biclique (No. %d)", num_maximal_bicliques);
                    ////     printf("\nL':");
                    ////     for (int i = 0; i < *NUM_L; i++)
                    ////         if (L[i] > lvl)
                    ////             printf(" %d", i);
                    ////     printf("\nR':");
                    ////     for (int i = 0; i < *R_lp_nxt; i++)
                    ////         printf(" %d", R[i]);
                    ////     printf("\n");
                    //// }

                    // save maximal bicliques
                    //// Biclique new_maximal_bicliques;
                    //// for (int i = 0; i < *L_lp_nxt; i++)
                    ////     new_maximal_bicliques.L.insert(L[i]);
                    //// for (int i = 0; i < *R_lp_nxt; i++)
                    ////     new_maximal_bicliques.R.insert(R[i]);
                    //// maximal_bicliques.push_back(new_maximal_bicliques);

                    // if (++num_maximal_bicliques > 0)
                    //     printf("blk %d : %d\n", blockIdx.x, num_maximal_bicliques);
                    printf("\33[%d;%dH%d\n", blockIdx.x / 10 + 1, (blockIdx.x % 10) * 10 + 1, ++num_maximal_bicliques);

                    // if P' ≠ ∅ then
                    if (*P_lp_nxt != 0) {
                        // biclique_find(G, L', R', P', Q');
                        //// printf("\n往 下 安安");
                        lvl++;
                        is_recursive = true;
                    }

                }

                __syncthreads();
            
                CLK(6);

                if (is_recursive)
                    break;

            }
            else {
                //// printf("\n不安安");
            }

            if (!threadIdx.x) {

                // Q ← Q ∪ {x};
                Q_rm[*Q_lp_cur] = INF;
                Q[(*Q_lp_cur)++] = *x_cur;
                //// printf("\n往 右 安安");

            }

        }
        
        __syncthreads();

        // printf("tid: %d, lvl: %d\n", tid, lvl);
        // 感覺這邊break完之後不用做?
        if (!threadIdx.x) {

            if (!is_recursive) {
                if (lvl--) {
                    Q_rm[Q_lp[lvl]] = INF;
                    Q[Q_lp[lvl]++] = x[lvl];
                }
                //// printf("\n往 上 安安");
                //// printf("\n往 右 安安");
            }

        }

        __syncthreads();

    }

    grid.sync();
    
    if (!threadIdx.x) {
        //// printf("\nBlock: %d find %d maximal bicliques.\n", blockIdx.x, num_maximal_bicliques);
        atomicAdd(&total_bic, num_maximal_bicliques);
        for (int i = 0; i < 10; i++) {
            clk[i] >>= 21;
            atomicAdd(&(g_clk[i]), (int)clk[i]);
        }
    }
    grid.sync();
    if (!tid) {
        printf("\33[%d;1Htotal maximal bicliques : %d\n", (gridDim.x + (10-1)) / 10 + 1, total_bic);
        printf("Time:");
        for (int i = 0; i < 10; i++)
            printf(" %d", g_clk[i]);
        printf("\n");
    }
    grid.sync();
}

int main(int argc, char* argv[])
{
    string str_dataset = argv[1];
    cout << str_dataset.substr(str_dataset.rfind('/')+1) << "\n";

    Node *node;
	int *edge;
    int *NUM_L, *NUM_R, *NUM_EDGES, _;
    // MBE
    int *u2L, *L, *R, *P, *Q;
    int *x, *L_lp, *R_lp, *P_lp, *Q_lp;
    int *Q_rm;
    // MBE_82
    int *g_u2L, *g_L, *g_R, *g_P, *g_Q;
    int *g_x, *g_L_lp, *g_R_lp, *g_P_lp, *g_Q_lp;
    int *g_Q_rm;
    hipMallocManaged(&NUM_EDGES, sizeof(int));
    hipMallocManaged(&NUM_L    , sizeof(int));
    hipMallocManaged(&NUM_R    , sizeof(int));

    ifstream fin;
    fin.open(argv[1]);
    fin >> *NUM_R >> *NUM_L >> *NUM_EDGES;
    hipMallocManaged(&node, sizeof(Node)*(*NUM_R    ));
    hipMallocManaged(&edge, sizeof(int )*(*NUM_EDGES));
    for (int i = 0; i < *NUM_R    ; i++) fin >> node[i].start >> node[i].length;
    for (int i = 0; i < *NUM_EDGES; i++) fin >> edge[i] >> _;
    fin.close();

    int numBlocksPerSM;
    int numThreads = NUM_THDS;
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_MBE, numThreads, 0);
    int numBlocks_max = deviceProp.multiProcessorCount * numBlocksPerSM;
    int numBlocks = NUM_BLKS <= numBlocks_max ? NUM_BLKS : numBlocks_max;
    dim3 num_blocks_MBE(1, 1, 1);
    dim3 num_blocks_MBE_82(numBlocks, 1, 1);
    dim3 block_size(numThreads, 1, 1);

    // MBE
    hipMallocManaged(&u2L , sizeof(int)*(*NUM_L)); my_memset_order(u2L, 0, *NUM_L);
    hipMallocManaged(&L   , sizeof(int)*(*NUM_L)); my_memset_order(L  , 0, *NUM_L);
    hipMallocManaged(&R   , sizeof(int)*(*NUM_R)); my_memset_order(R  , 0, *NUM_R);
    hipMallocManaged(&P   , sizeof(int)*(*NUM_R)); my_memset_order(P  , 0, *NUM_R);
    hipMallocManaged(&Q   , sizeof(int)*(*NUM_R)); my_memset_order(Q  , 0, *NUM_R);
    hipMallocManaged(&x   , sizeof(int)*(*NUM_R)); my_memset(x   ,     -1, *NUM_R);
    hipMallocManaged(&L_lp, sizeof(int)*(*NUM_R)); my_memset(L_lp, *NUM_L, *NUM_R);
    hipMallocManaged(&R_lp, sizeof(int)*(*NUM_R)); my_memset(R_lp,      0, *NUM_R);
    hipMallocManaged(&P_lp, sizeof(int)*(*NUM_R)); my_memset(P_lp, *NUM_R, *NUM_R);
    hipMallocManaged(&Q_lp, sizeof(int)*(*NUM_R)); my_memset(Q_lp,      0, *NUM_R);
    hipMallocManaged(&Q_rm, sizeof(int)*(*NUM_R)); my_memset(Q_rm,    INF, *NUM_R);
    // MBE_82
    hipMallocManaged(&g_u2L , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )  g_u2L[i] =  u2L[i % (*NUM_L)];
    hipMallocManaged(&g_L   , sizeof(int)*(*NUM_L)*numBlocks); for (int i = numBlocks * (*NUM_L); i-- > 0; )    g_L[i] =    L[i % (*NUM_L)];
    hipMallocManaged(&g_R   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_R[i] =    R[i % (*NUM_R)];
    hipMallocManaged(&g_P   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_P[i] =    P[i % (*NUM_R)];
    hipMallocManaged(&g_Q   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_Q[i] =    Q[i % (*NUM_R)];
    hipMallocManaged(&g_x   , sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; )    g_x[i] =    x[i % (*NUM_R)];
    hipMallocManaged(&g_L_lp, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_L_lp[i] = L_lp[i % (*NUM_R)];
    hipMallocManaged(&g_R_lp, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_R_lp[i] = R_lp[i % (*NUM_R)];
    hipMallocManaged(&g_P_lp, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_P_lp[i] = P_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_lp, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_Q_lp[i] = Q_lp[i % (*NUM_R)];
    hipMallocManaged(&g_Q_rm, sizeof(int)*(*NUM_R)*numBlocks); for (int i = numBlocks * (*NUM_R); i-- > 0; ) g_Q_rm[i] = Q_rm[i % (*NUM_R)];

    void *kernelArgs_MBE[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node, &edge, &u2L, &L, &R, &P, &Q, &x, &L_lp, &R_lp, &P_lp, &Q_lp};
    void *kernelArgs_MBE_82[] = {&NUM_L, &NUM_R, &NUM_EDGES, &node, &edge, &g_u2L, &g_L, &g_R, &g_P, &g_Q, &g_Q_rm, &g_x, &g_L_lp, &g_R_lp, &g_P_lp, &g_Q_lp};

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int stat;
    int mode = NUM_BLKS;
    if (mode == -2)
        maximal_bic_enum_set(NUM_L, NUM_R, NUM_EDGES, node, edge, u2L, L, R, P, Q, x, L_lp, R_lp, P_lp, Q_lp);
    else if (mode == -1)
        maximal_bic_enum(NUM_L, NUM_R, NUM_EDGES, node, edge, u2L, L, R, P, Q, x, L_lp, R_lp, P_lp, Q_lp);
    else if (mode == 0) {
        hipLaunchCooperativeKernel((void*)CUDA_MBE, num_blocks_MBE, block_size, kernelArgs_MBE);
    }
    else {
        cout << "\33[2J\n";
        stat = hipLaunchCooperativeKernel((void*)CUDA_MBE_82, num_blocks_MBE_82, block_size, kernelArgs_MBE_82);
    }
    
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    cout << "status: " << stat << "\n";
    printf("Running time: %f secs\n", time/1000);

    hipFree(node);
    hipFree(edge);
    hipFree(NUM_L);
    hipFree(NUM_R);
    hipFree(NUM_EDGES);
    // MBE
    hipFree(u2L);
    hipFree(L);
    hipFree(R);
    hipFree(P);
    hipFree(Q);
    hipFree(x);
    hipFree(L_lp);
    hipFree(R_lp);
    hipFree(P_lp);
    hipFree(Q_lp);
    // MBE_82
    hipFree(g_u2L);
    hipFree(g_L);
    hipFree(g_R);
    hipFree(g_P);
    hipFree(g_Q);
    hipFree(g_x);
    hipFree(g_L_lp);
    hipFree(g_R_lp);
    hipFree(g_P_lp);
    hipFree(g_Q_lp);
}