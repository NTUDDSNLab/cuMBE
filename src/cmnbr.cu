#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <bits/stdc++.h>
#include <hip/hip_cooperative_groups.h>
#include <sys/ioctl.h>
using namespace std;
using namespace std::chrono;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define NUM_THDS 512 // argv[2]
#define FEAT_DIM 128 // argv[3]
#define FEAT_AVG   0
#define FEAT_STD  10
#define DEF_STEP  16 // argv[4]

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

struct EdgePair
{
	int row;
	int col;
    EdgePair(int row_, int col_): row(row_), col(col_) {}
};

typedef struct
{
    int num_rows;
    int num_cols;
    unordered_set<int> row;
    unordered_set<int> col;
} Biclique;

typedef struct
{
    int row_end, r;
    int col_end, c;
    int *row;
    int *col;
} Candidate;

random_device rd;
mt19937 en(rd());
uniform_int_distribution<unsigned long long> rand_64;

char t_ms_idx = 0; long long t_ms_start = 0, t_ms_end = 0; vector<long long> t_ms(12, 0);
void mark_time_ms(short new_init_idx) {
    // t_ms_end = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
    t_ms_end = clock();
    if (new_init_idx >= 0) t_ms_idx = new_init_idx;
    else t_ms[t_ms_idx++] += t_ms_end - t_ms_start;
    t_ms_start = t_ms_end;
    return;
}

char choose_op(int num_rows_cand, int num_cols_cand, int num_rows_bic, int num_cols_bic) {
    // cout << num_rows_cand << ' ' << num_cols_cand << ' ' << num_rows_bic << ' ' << num_cols_bic << ' ';
    if ((num_rows_cand == 0 && num_rows_bic == 1) || (num_cols_cand == 0 && num_cols_bic == 1) || (num_rows_cand | num_cols_cand) == 0) return 'x';
    long long prob_rows = 1, prob_cols = 1;
    if (num_rows_bic != 1 || num_cols_bic != 1) {
        prob_rows = (num_rows_cand != 0) * (num_cols_bic - 1);
        prob_cols = (num_cols_cand != 0) * (num_rows_bic - 1) * 2;
    }
    uniform_int_distribution<> rand_cand(0, prob_rows + prob_cols - 1);
    return (rand_cand(en) < prob_rows ? 'r' : 'c');
}

__global__ void CUDA_GNN_GEN_FEATS(double *feat_i, long long *NUM_NODES, long long *NUM_FEATS)
{
    for(long long i = blockIdx.x; i < *NUM_NODES; i += gridDim.x)
        for(long long j = threadIdx.x; j < *NUM_FEATS; j += blockDim.x)
            feat_i[(*NUM_FEATS) * i + j] = threadIdx.x + blockIdx.x;
}

__global__ void CUDA_CMNBRGNN_KERNEL(Node *node_bic, int *edge_bic, double *feat_bic, Node *node, int *edge, double *feat_i, double *feat_o, int *dest, int *BIC_STEP, long long *NUM_BICS, int *NODE_STEP, long long *NUM_NODES, long long *NUM_FEATS_i, long long *NUM_FEATS_o)
{
    grid_group grid = this_grid();
    int lid = threadIdx.x & 0x1f;
    long long did, sid;
    double result_tmp;

    *dest = 0;
    grid.sync();
    
    if (lid == 0) did = atomicAdd(dest, *BIC_STEP);
    did = __shfl_sync(0xffffffff, did, 0, 32);
    for (; did < *NUM_BICS; ){
        // if (lid == 0) printf("%d\n", did);
        for (int did_end = min(did + *BIC_STEP, *NUM_BICS); did < did_end; did++) {
            for (long long fid = lid; fid < *NUM_FEATS_i; fid += 32) {
                result_tmp = 0;
                for (int eid = node_bic[did].start, eid_end = eid + node_bic[did].length; eid < eid_end; eid++) {
                    sid = edge_bic[eid];
                    result_tmp += feat_i[sid * (*NUM_FEATS_i) + fid];
                }
                feat_bic[did * (*NUM_FEATS_i) + fid] = result_tmp;
        }   }
        if (lid == 0) did = atomicAdd(dest, *BIC_STEP);
        did = __shfl_sync(0xffffffff, did, 0, 32);
    }
    grid.sync();

    *dest = 0;
    grid.sync();
    
    if (lid == 0) did = atomicAdd(dest, *NODE_STEP);
    did = __shfl_sync(0xffffffff, did, 0, 32);
    for (; did < *NUM_NODES; ){
        for (int did_end = min(did + *NODE_STEP, *NUM_NODES); did < did_end; did++) {
            for (long long fid = lid; fid < *NUM_FEATS_i; fid += 32) {
                result_tmp = 0;
                for (int eid = node[did].start, eid_end = eid + node[did].length; eid < eid_end; eid++) {
                    sid = edge[eid];
                    // if (did * (*NUM_FEATS_i) + fid == 118479) printf("%d %d %d\n", eid, fid, feat_i[sid * (*NUM_FEATS_i) + fid]);
                    result_tmp += sid >= 0 ? feat_i[sid * (*NUM_FEATS_i) + fid] : feat_bic[~sid * (*NUM_FEATS_i) + fid];
                }
                feat_o[did * (*NUM_FEATS_i) + fid] = result_tmp;
        }   }
        if (lid == 0) did = atomicAdd(dest, *NODE_STEP);
        did = __shfl_sync(0xffffffff, did, 0, 32);
    }
    grid.sync();
}

int main(int argc, char* argv[])
{
    string str_dataset = argv[1];
    // printf("\033[0;1;33m");
    cout << str_dataset.substr(str_dataset.rfind('/')+1) << "\n";
    // printf("\033[0;1m");
	ifstream fin;
    int _, *NUM_EDGES, SOURCE, *NODE_STEP, *NUM_BIC_EDGES, *BIC_STEP;
    long long *NUM_NODES, *NUM_FEATS, *NUM_BICS;
    hipMallocManaged(&NODE_STEP, sizeof(int));
    hipMallocManaged(&NUM_EDGES, sizeof(int));
    hipMallocManaged(&NUM_NODES, sizeof(long long));
    hipMallocManaged(&NUM_FEATS, sizeof(long long));
    hipMallocManaged(&BIC_STEP, sizeof(int));
    hipMallocManaged(&NUM_BICS, sizeof(long long));
    hipMallocManaged(&NUM_BIC_EDGES, sizeof(int));

    *NUM_FEATS = (argc > 3) ? atoi(argv[3]) : FEAT_DIM;
    *NODE_STEP = (argc > 4) ? atoi(argv[4]) : DEF_STEP;
    *BIC_STEP = 1;

    fin.open(argv[5]);
    fin >> *NUM_NODES >> *NUM_EDGES;

	Node* node;
	int* edge;
    double* feat_i;
    double* feat_o;
    hipMallocManaged(&node, sizeof(Node)*(*NUM_NODES));
    hipMallocManaged(&edge, sizeof(int)*(*NUM_EDGES));
    hipMallocManaged(&feat_i, sizeof(double)*((*NUM_NODES)*(*NUM_FEATS)));
    hipMallocManaged(&feat_o, sizeof(double)*((*NUM_NODES)*(*NUM_FEATS)));

    for(int i=0;i<*NUM_NODES;i++) fin >> node[i].start >> node[i].length;
    for(int i=0;i<*NUM_EDGES;i++) fin >> edge[i];
    fin >> *NUM_BICS >> *NUM_BIC_EDGES;

    Node *node_c;
	int *edge_c;
    double* feat_c;
    hipMallocManaged(&node_c, sizeof(Node)*(*NUM_BICS));     // biclique list (node)
    hipMallocManaged(&edge_c, sizeof(int)*(*NUM_BIC_EDGES)); // biclique list (edge)
    hipMallocManaged(&feat_c, sizeof(double)*((*NUM_BICS)*(*NUM_FEATS)));

    for(int i=0;i<*NUM_BICS;i++) fin >> node_c[i].start >> node_c[i].length;
    for(int i=0;i<*NUM_BIC_EDGES;i++) fin >> edge_c[i];
    fin.close();
    
    cout << "Nodes: " << *NUM_NODES << "\n";
    cout << "Edges: " << *NUM_EDGES << "\n";
    cout << "Feats: " << *NUM_FEATS << "\n";
    cout << "Step: "  << *NODE_STEP << "\n";

    int numBlocksPerSM = 1;
    int numThreads = (argc > 2) ? atoi(argv[2]) : NUM_THDS;
    int* dest;
    hipMallocManaged(&dest, sizeof(int));
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_GNN_GEN_FEATS, numThreads, 0);
    dim3 num_blocks_GNN_GEN_FEATS(deviceProp.multiProcessorCount * numBlocksPerSM, 1, 1);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSM, CUDA_CMNBRGNN_KERNEL, numThreads, 0);
    dim3 num_blocks_CMNBRGNN_KERNEL(deviceProp.multiProcessorCount * numBlocksPerSM, 1, 1);
    dim3 block_size(numThreads, 1, 1);
    void *kernelArgs_GNN_GEN_FEATS[] = {&feat_i, &NUM_NODES, &NUM_FEATS};
    void *kernelArgs_CMNBRGNN_KERNEL[] = {&node_c, &edge_c, &feat_c, &node, &edge, &feat_i, &feat_o, &dest, &BIC_STEP, &NUM_BICS, &NODE_STEP, &NUM_NODES, &NUM_FEATS, &NUM_FEATS};
    cout << "block_size: " << numThreads << "\n";
    cout << "num_blocks_GNN_GEN_FEATS:   " << num_blocks_GNN_GEN_FEATS.x   << "\n";
    cout << "num_blocks_CMNBRGNN_KERNEL: " << num_blocks_CMNBRGNN_KERNEL.x << "\n";

    // cout << "Kernel Start\n";
    hipLaunchCooperativeKernel((void*)CUDA_GNN_GEN_FEATS, num_blocks_GNN_GEN_FEATS, block_size, kernelArgs_GNN_GEN_FEATS);
    hipDeviceSynchronize();
    
    // cout << "Prefetch\n";
    hipMemPrefetchAsync(node_c, sizeof(Node)*(*NUM_BICS), device, NULL);
    hipMemPrefetchAsync(edge_c, sizeof(int)*(*NUM_BIC_EDGES), device, NULL);
    hipMemPrefetchAsync(feat_c, sizeof(double)*((*NUM_BICS)*(*NUM_FEATS)), device, NULL);
    hipMemPrefetchAsync(node, sizeof(Node)*(*NUM_NODES), device, NULL);
    hipMemPrefetchAsync(edge, sizeof(int)*(*NUM_EDGES), device, NULL);
    hipMemPrefetchAsync(feat_i, sizeof(double)*((*NUM_NODES)*(*NUM_FEATS)), device, NULL);
    hipMemPrefetchAsync(feat_o, sizeof(double)*((*NUM_NODES)*(*NUM_FEATS)), device, NULL);
    hipMemPrefetchAsync(dest, sizeof(int), device, NULL);
    hipMemPrefetchAsync(BIC_STEP, sizeof(int), device, NULL);
    hipMemPrefetchAsync(NUM_BICS, sizeof(long long), device, NULL);
    hipMemPrefetchAsync(NODE_STEP, sizeof(int), device, NULL);
    hipMemPrefetchAsync(NUM_NODES, sizeof(long long), device, NULL);
    hipMemPrefetchAsync(NUM_FEATS, sizeof(long long), device, NULL);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < 1000; i++) {
        hipLaunchCooperativeKernel((void*)CUDA_CMNBRGNN_KERNEL, num_blocks_CMNBRGNN_KERNEL, block_size, kernelArgs_CMNBRGNN_KERNEL);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    // cout << "Kernel End\n";

    fin.open(argv[1]);
    fin >> *NUM_NODES >> *NUM_EDGES >> SOURCE;
    for(int i=0;i<*NUM_NODES;i++) fin >> node[i].start >> node[i].length;
    hipFree(edge);
    hipMallocManaged(&edge, sizeof(int)*(*NUM_EDGES));
    for(int i=0;i<*NUM_EDGES;i++) fin >> edge[i] >> _;
    fin.close();

    long long num_errors = 0;
    for (long long i = 0; i < *NUM_NODES; i++)
        for (long long j = 0; j < *NUM_FEATS; j++) {
            double result_tmp = 0;
            for (int k = node[i].start, k_end = k + node[i].length; k < k_end; k++) {
                long long sid = edge[k];
                result_tmp += feat_i[sid * (*NUM_FEATS) + j];
            }
            if (result_tmp != feat_o[i * (*NUM_FEATS) + j]) {
                num_errors++;
            }
        }

    cout << "- Time  = " << time << "ms" << "\n";
    cout << "- Error = " << num_errors << '/' << (*NUM_NODES)*(*NUM_FEATS) << "\n";

    // printf("\033[0m");
    hipFree(BIC_STEP);
    hipFree(NODE_STEP);
    hipFree(NUM_EDGES);
    hipFree(NUM_NODES);
    hipFree(NUM_FEATS);
    hipFree(NUM_BICS);
    hipFree(NUM_BIC_EDGES);
    hipFree(node);
    hipFree(edge);
    hipFree(node_c);
    hipFree(edge_c);
    hipFree(feat_c);
    hipFree(feat_i);
    hipFree(feat_o);
    hipFree(dest);
}